#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for texture-memory interpolation based projection
 *
 * This file has the necesary fucntiosn to perform X-ray CBCT projection
 * operation given a geaometry, angles and image. It uses the 3D texture
 * memory linear interpolation to uniformily sample a path to integrate the
 * X-rays.
 *
 * CODE by       Ander Biguri
 *               Sepideh Hatamikia (arbitrary rotation)
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */






#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "ray_interpolated_projection.hpp"
#include "mex.h"
#include <math.h>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                hipDeviceReset();\
                        mexErrMsgIdAndTxt("TIGRE:Ax:interpolated",hipGetErrorString(__err));\
        } \
} while (0)
    
    
// Declare the texture reference.
    
#define MAXTREADS 1024
#define PROJ_PER_BLOCK 9
#define PIXEL_SIZE_BLOCK 9
    /*GEOMETRY DEFINITION
     *
     *                Detector plane, behind
     *            |-----------------------------|
     *            |                             |
     *            |                             |
     *            |                             |
     *            |                             |
     *            |      +--------+             |
     *            |     /        /|             |
     *   A Z      |    /        / |*D           |
     *   |        |   +--------+  |             |
     *   |        |   |        |  |             |
     *   |        |   |     *O |  +             |
     *    --->y   |   |        | /              |
     *  /         |   |        |/               |
     * V X        |   +--------+                |
     *            |-----------------------------|
     *
     *           *S
     *
     *
     *
     *
     *
     **/
    void CreateTextureInterp(int num_devices,const float* imagedata,Geometry geo,hipArray** d_cuArrTex, hipTextureObject_t *texImage,bool allocate);
__constant__ Point3D projParamsArrayDev[4*PROJ_PER_BLOCK];  // Dev means it is on device
__constant__ float projFloatsArrayDev[2*PROJ_PER_BLOCK];  // Dev means it is on device


__global__ void vecAddInPlaceInterp(float *a, float *b, unsigned long  n)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (idx < n)
        a[idx] = a[idx] + b[idx];
}


template<bool sphericalrotation>
        __global__ void kernelPixelDetector( Geometry geo,
        float* detector,
        const int currProjSetNumber,
        const int totalNoOfProjections,
        hipTextureObject_t tex){
    
    unsigned long  y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long  x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long projNumber=threadIdx.z;
    
    
    if ((x>= geo.nDetecU) | (y>= geo.nDetecV)|  (projNumber>=PROJ_PER_BLOCK))
        return;
    
    size_t idx =  (size_t)(x  * geo.nDetecV + y)+ (size_t)projNumber*geo.nDetecV *geo.nDetecU ;
    int indAlpha = currProjSetNumber*PROJ_PER_BLOCK+projNumber;  // This is the ABSOLUTE projection number in the projection array
    
    if(indAlpha>=totalNoOfProjections)
        return;
    
    Point3D uvOrigin = projParamsArrayDev[4*projNumber];  // 6*projNumber because we have 6 Point3D values per projection
    Point3D deltaU = projParamsArrayDev[4*projNumber+1];
    Point3D deltaV = projParamsArrayDev[4*projNumber+2];
    Point3D source = projParamsArrayDev[4*projNumber+3];
    
    float DSO = projFloatsArrayDev[2*projNumber+0];
    float cropdist_init = projFloatsArrayDev[2*projNumber+1];
    
    
    
    /////// Get coordinates XYZ of pixel UV
    int pixelV = geo.nDetecV-y-1;
    int pixelU = x;
    
    
    
    float vectX,vectY,vectZ;
    Point3D P;
    P.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    P.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    P.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    
    // Length is the ray length in normalized space
    float length=__fsqrt_rd((source.x-P.x)*(source.x-P.x)+(source.y-P.y)*(source.y-P.y)+(source.z-P.z)*(source.z-P.z));
    //now legth is an integer of Nsamples that are required on this line
    length=ceilf(__fdividef(length,geo.accuracy));//Divide the directional vector by an integer
    vectX=__fdividef(P.x -source.x,length);
    vectY=__fdividef(P.y -source.y,length);
    vectZ=__fdividef(P.z -source.z,length);
    
    
//     //Integrate over the line
    float tx,ty,tz;
    float sum=0;
    float i;
    
    
    
//  Because I have no idea how to efficiently cutoff the legth path in 3D, a very upper limit is computed (see maxdistanceCuboid)
//  for the 3D case. However it would be bad to lose performance in the 3D case
//  TODO: can ge really improve this?
    if (sphericalrotation){
        if ((2*DSO/fminf(fminf(geo.dVoxelX,geo.dVoxelY),geo.dVoxelZ)+cropdist_init)/geo.accuracy  <   length)
            length=ceilf((2*DSO/fminf(fminf(geo.dVoxelX,geo.dVoxelY),geo.dVoxelZ)+cropdist_init)/geo.accuracy);
    }
    else{
        if ((2*DSO/fminf(geo.dVoxelX,geo.dVoxelY)+cropdist_init)/geo.accuracy  <   length)
            length=ceilf((2*DSO/fminf(geo.dVoxelX,geo.dVoxelY)+cropdist_init)/geo.accuracy);
    }
    
    
    //Length is not actually a length, but the amount of memreads with given accuracy ("samples per voxel")
    for (i=floorf(cropdist_init/geo.accuracy); i<=length; i=i+1){
        tx=vectX*i+source.x;
        ty=vectY*i+source.y;
        tz=vectZ*i+source.z;
        
        sum += tex3D<float>(tex, tx+0.5f, ty+0.5f, tz+0.5f); // this line is 94% of time.
    }
    
    float deltalength=sqrtf((vectX*geo.dVoxelX)*(vectX*geo.dVoxelX)+
            (vectY*geo.dVoxelY)*(vectY*geo.dVoxelY)+
            (vectZ*geo.dVoxelZ)*(vectZ*geo.dVoxelZ) );
    
    detector[idx]=sum*deltalength;
}



// legnth(angles)=3 x nagnles, as we have roll, pitch, yaw.
int interpolation_projection(float  *  img, Geometry geo, float** result,float const * const angles,int nangles){
    
    
    // Prepare for MultiGPU
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    cudaCheckErrors("Device query fail");
    if (deviceCount == 0) {
        mexErrMsgIdAndTxt("Ax:Interpolated_projection:GPUselect","There are no available device(s) that support CUDA\n");
    }
    //
    // CODE assumes
    // 1.-All available devices are usable by this code
    // 2.-All available devices are equal, they are the same machine (warning trhown)
    int dev;
    char * devicenames;
    hipDeviceProp_t deviceProp;
    
    for (dev = 0; dev < deviceCount; dev++) {
        hipSetDevice(dev);
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev>0){
            if (strcmp(devicenames,deviceProp.name)!=0){
                mexWarnMsgIdAndTxt("Ax:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed. \n Siddon_projection.cu line 275.");
                break;
            }
        }
        devicenames=deviceProp.name;
    }
    
    // Check free memory
    size_t mem_GPU_global;
    checkFreeMemory(deviceCount,&mem_GPU_global);
    
    size_t mem_image=(unsigned long long)geo.nVoxelX*(unsigned long long)geo.nVoxelY*(unsigned long long)geo.nVoxelZ*sizeof(float);
    size_t mem_proj =(unsigned long long)geo.nDetecU*(unsigned long long)geo.nDetecV * sizeof(float);
    
    // Does everything fit in the GPUs?
    bool fits_in_memory=false;
    unsigned int splits=1;
    Geometry * geoArray;
    
    
    if (mem_image+2*PROJ_PER_BLOCK*mem_proj<mem_GPU_global){// yes it does
        fits_in_memory=true;
        geoArray=(Geometry*)malloc(sizeof(Geometry));
        geoArray[0]=geo;
    }
    else{// Nope nope.
        fits_in_memory=false; // Oh dear.
        // approx free memory we have. We already have left some extra 10% free for internal stuff
        // we need a second projection memory to combine multi-GPU stuff.
        size_t mem_free=mem_GPU_global-4*PROJ_PER_BLOCK*mem_proj;
        
        
        splits=mem_image/mem_free+1;// Ceil of the truncation
        geoArray=(Geometry*)malloc(splits*sizeof(Geometry));
        splitImageInterp(splits,geo,geoArray,nangles);
    }
    
    // Allocate auiliary memory for projections on the GPU to accumulate partial resutsl
    float ** dProjection_accum;
    size_t num_bytes_proj = PROJ_PER_BLOCK*geo.nDetecU*geo.nDetecV * sizeof(float);
    if (!fits_in_memory){
        dProjection_accum=(float**)malloc(2*deviceCount*sizeof(float*));
        for (dev = 0; dev < deviceCount; dev++) {
            hipSetDevice(dev);
            for (int i = 0; i < 2; ++i){
                hipMalloc((void**)&dProjection_accum[dev*2+i], num_bytes_proj);
                hipMemset(dProjection_accum[dev*2+i],0,num_bytes_proj);
                cudaCheckErrors("cudaMallocauxiliarty projections fail");
            }
        }
    }
    
    // This is happening regarthless if the image fits on memory
    float** dProjection=(float**)malloc(2*deviceCount*sizeof(float*));
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        
        for (int i = 0; i < 2; ++i){
            hipMalloc((void**)&dProjection[dev*2+i],   num_bytes_proj);
            hipMemset(dProjection[dev*2+i]  ,0,num_bytes_proj);
            cudaCheckErrors("hipMalloc projections fail");
        }
    }
    
    
    
    
    //Pagelock memory for syncronous copy.
    // Lets try to make the host memory pinned:
    // We laredy queried the GPU and assuemd they are the same, thus shoudl have the same attributes.
    int isHostRegisterSupported;
    hipDeviceGetAttribute(&isHostRegisterSupported,hipDeviceAttributeHostRegisterSupported,0);
    // empirical testing shows that when the image split is smaller than 1 (also implies the image is not very big), the time to
    // pin the memory is greater than the lost time in Syncronously launching the memcpys. This is only worth it when the image is too big.
    if (isHostRegisterSupported & splits>1){
        hipHostRegister(img, (size_t)geo.nVoxelX*(size_t)geo.nVoxelY*(size_t)geo.nVoxelZ*(size_t)sizeof(float),hipHostRegisterPortable);
    }
    
    
    
    
    Point3D source, deltaU, deltaV, uvOrigin;
    
    Point3D* projParamsArrayHost;
    hipHostMalloc((void**)&projParamsArrayHost,4*PROJ_PER_BLOCK*sizeof(Point3D));
    float* projFloatsArrayHost;
    hipHostMalloc((void**)&projFloatsArrayHost,2*PROJ_PER_BLOCK*sizeof(float));
    
    
    // Create Streams for overlapping memcopy and compute
    int nStream_device=2;
    int nStreams=deviceCount*nStream_device;
    hipStream_t* stream=(hipStream_t*)malloc(nStreams*sizeof(hipStream_t));
    
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        for (int i = 0; i < nStream_device; ++i){
            hipStreamCreate(&stream[i+dev*nStream_device]);
            
        }
    }
    cudaCheckErrors("Stream creation fail");
    int nangles_device=(nangles+deviceCount-1)/deviceCount;
    int nangles_last_device=(nangles-(deviceCount-1)*nangles_device);
    unsigned int noOfKernelCalls = (nangles_device+PROJ_PER_BLOCK-1)/PROJ_PER_BLOCK;  // We'll take care of bounds checking inside the loop if nalpha is not divisible by PROJ_PER_BLOCK
    unsigned int last_device_blocks= (nangles_last_device+PROJ_PER_BLOCK-1)/PROJ_PER_BLOCK; // we will use this in the memory management.
    int projection_this_block;

    
    hipTextureObject_t *texImg = new hipTextureObject_t[deviceCount];
    hipArray **d_cuArrTex = new hipArray*[deviceCount];
    for (unsigned int sp=0;sp<splits;sp++){
        
        // Create texture objects for all GPUs
        
        
        size_t linear_idx_start;
        //First one shoudl always be  the same size as all the rest but the last
        linear_idx_start= (size_t)sp*(size_t)geoArray[0].nVoxelX*(size_t)geoArray[0].nVoxelY*(size_t)geoArray[0].nVoxelZ;
        CreateTextureInterp(deviceCount,&img[linear_idx_start],geoArray[sp],d_cuArrTex,texImg,!sp);
        cudaCheckErrors("Texture object creation fail");
        
        
        int divU,divV;
        divU=PIXEL_SIZE_BLOCK;
        divV=PIXEL_SIZE_BLOCK;
        dim3 grid((geoArray[sp].nDetecU+divU-1)/divU,(geoArray[0].nDetecV+divV-1)/divV,1);
        dim3 block(divU,divV,PROJ_PER_BLOCK);
        
        unsigned int proj_global;
        unsigned int i;
        float maxdist;
        // Now that we have prepared the image (piece of image) and parameters for kernels
        // we project for all angles.
        for ( i=0; i<noOfKernelCalls; i++){
            for (dev=0;dev<deviceCount;dev++){
                float is_spherical=0;
                hipSetDevice(dev);
                
                for(unsigned int j=0; j<PROJ_PER_BLOCK; j++){
                    proj_global=(i*PROJ_PER_BLOCK+j)+dev*nangles_device;
                    if (proj_global>=nangles)
                        break;
                     if ((i*PROJ_PER_BLOCK+j)>=nangles_device)
                        break;
                    geo.alpha=angles[proj_global*3];
                    geo.theta=angles[proj_global*3+1];
                    geo.psi  =angles[proj_global*3+2];
                    
                    is_spherical+=abs(geo.theta)+abs(geo.psi);
                    
                    //precomute distances for faster execution
                    maxdist=maxdistanceCuboid(geo,proj_global);
                    //Precompute per angle constant stuff for speed
                    computeDeltas(geo, proj_global, &uvOrigin, &deltaU, &deltaV, &source);
                    //Ray tracing!
                    projParamsArrayHost[4*j]=uvOrigin;		// 6*j because we have 6 Point3D values per projection
                    projParamsArrayHost[4*j+1]=deltaU;
                    projParamsArrayHost[4*j+2]=deltaV;
                    projParamsArrayHost[4*j+3]=source;
                    
                    projFloatsArrayHost[2*j]=geo.DSO[proj_global];
                    projFloatsArrayHost[2*j+1]=floor(maxdist);
                }
                
                hipMemcpyToSymbolAsync(HIP_SYMBOL(projParamsArrayDev), projParamsArrayHost, sizeof(Point3D)*4*PROJ_PER_BLOCK,0,hipMemcpyHostToDevice,stream[dev*nStream_device]);
                hipMemcpyToSymbolAsync(HIP_SYMBOL(projFloatsArrayDev), projFloatsArrayHost, sizeof(float)*2*PROJ_PER_BLOCK,0,hipMemcpyHostToDevice,stream[dev*nStream_device]);
                hipStreamSynchronize(stream[dev*nStream_device]);
                
                
                //TODO: we could do this around X and Y axis too, but we would need to compute the new axis of rotation (not possible to know from jsut the angles)
                if (!is_spherical){
                    kernelPixelDetector<false><<<grid,block,0,stream[dev*nStream_device]>>>(geoArray[sp],dProjection[(i%2)+dev*2],i,nangles_device,texImg[dev]);
                }
                else{
                    kernelPixelDetector<true> <<<grid,block,0,stream[dev*nStream_device]>>>(geoArray[sp],dProjection[(i%2)+dev*2],i,nangles_device,texImg[dev]);
                }
            }
            
            
            // Now that the computation is happening, we need to either prepare the memory for
            // combining of the projections (splits>1) and start removing previous results.
            
            
            // If our image does not fit in memory then we need to make sure we accumulate previous results too.
            // This is done in 2 steps: 
            // 1)copy previous results back into GPU 
            // 2)accumulate with current results
            // The code to take them out is the same as when there are no splits needed
            if( !fits_in_memory&&sp>0)
            {
                // 1) grab previous results and put them in the auxiliary variable dProjection_accum
                for (dev = 0; dev < deviceCount; dev++)
                {
                    hipSetDevice(dev);
                    //Global index of FIRST projection on this set on this GPU
                    proj_global=i*PROJ_PER_BLOCK+dev*nangles_device;
                    if(proj_global>=nangles) 
                        break;

                    // Unless its the last projection set, we have PROJ_PER_BLOCK angles. Otherwise...
                    if(i+1==noOfKernelCalls) //is it the last block?
                        projection_this_block=min(nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK, //the remaining angles that this GPU had to do (almost never PROJ_PER_BLOCK)
                                                  nangles-proj_global);                              //or whichever amount is left to finish all (this is for the last GPU)
                    else
                        projection_this_block=PROJ_PER_BLOCK;

                    hipMemcpyAsync(dProjection_accum[(i%2)+dev*2], result[proj_global], projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyHostToDevice,stream[dev*2+1]);
                }
                //  2) take the results from current compute call and add it to the code in execution.
                for (dev = 0; dev < deviceCount; dev++)
                {
                    hipSetDevice(dev);
                    //Global index of FIRST projection on this set on this GPU
                    proj_global=i*PROJ_PER_BLOCK+dev*nangles_device;
                    if(proj_global>=nangles) 
                        break;

                    // Unless its the last projection set, we have PROJ_PER_BLOCK angles. Otherwise...
                    if(i+1==noOfKernelCalls) //is it the last block?
                        projection_this_block=min(nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK, //the remaining angles that this GPU had to do (almost never PROJ_PER_BLOCK)
                                                  nangles-proj_global);                              //or whichever amount is left to finish all (this is for the last GPU)
                    else
                        projection_this_block=PROJ_PER_BLOCK;

                    hipStreamSynchronize(stream[dev*2+1]); // wait until copy is finished
                    vecAddInPlaceInterp<<<(geo.nDetecU*geo.nDetecV*projection_this_block+MAXTREADS-1)/MAXTREADS,MAXTREADS,0,stream[dev*2]>>>(dProjection[(i%2)+dev*2],dProjection_accum[(i%2)+dev*2],(unsigned long)geo.nDetecU*geo.nDetecV*projection_this_block);
                }
            } // end accumulation case, where the image needs to be split 

            // Now, lets get out the projections from the previous execution of the kernels.
            if (i>0){
                for (dev = 0; dev < deviceCount; dev++)
                {
                    hipSetDevice(dev);
                    //Global index of FIRST projection on previous set on this GPU
                    proj_global=(i-1)*PROJ_PER_BLOCK+dev*nangles_device;
                    //Unless it is the last (handled separately later), all blocks are full.
                    projection_this_block=PROJ_PER_BLOCK;
                    hipMemcpyAsync(result[proj_global], dProjection[(int)(!(i%2))+dev*2],  projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyDeviceToHost,stream[dev*2+1]);
                }
            }
            // Make sure Computation on kernels has finished before we launch the next batch.
            for (dev = 0; dev < deviceCount; dev++){
                hipSetDevice(dev);
                hipStreamSynchronize(stream[dev*2]);
            }
        } // End noOfKernelCalls (i) loop.
        
        // We still have the last set of projections to get out of all GPUs
        //Note: noOfKernelCalls==i
        for (dev = 0; dev < deviceCount; dev++)
        {
            hipSetDevice(dev);
            //Global index of FIRST projection on this set on this GPU
            proj_global=(noOfKernelCalls-1)*PROJ_PER_BLOCK+dev*nangles_device;
            if(proj_global>=nangles) 
                break;
            // How many projections are left here?
            projection_this_block=min(nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK, //the remaining angles that this GPU had to do (almost never PROJ_PER_BLOCK)
                                      nangles-proj_global);                              //or whichever amount is left to finish all (this is for the last GPU)

            hipDeviceSynchronize(); //Not really necesary, but just in case, we los nothing. 
            cudaCheckErrors("Error at copying the last set of projections out (or in the previous copy)");
            hipMemcpyAsync(result[proj_global], dProjection[(int)(!(noOfKernelCalls%2))+dev*2], projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyDeviceToHost,stream[dev*2+1]);
        }
        // Make sure everyone has done their bussiness before the next image split:
        hipDeviceSynchronize();
    } // End image split loop.
    
    cudaCheckErrors("Main loop  fail");
    ///////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipDestroyTextureObject(texImg[dev]);
        hipFreeArray(d_cuArrTex[dev]);
    }
    // Freeing Stage
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipFree(dProjection[dev*2]);
        hipFree(dProjection[dev*2+1]);
        
    }
    free(dProjection);
    
    if(!fits_in_memory){
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipFree(dProjection_accum[dev*2]);
            hipFree(dProjection_accum[dev*2+1]);
            
        }
        free(dProjection_accum);
    }
    freeGeoArray(splits,geoArray);
    hipHostFree(projParamsArrayHost);
    
    
    for (int i = 0; i < nStreams; ++i)
        hipStreamDestroy(stream[i]) ;
    
    if (isHostRegisterSupported & splits>1){
        hipHostUnregister(img);
    }
    cudaCheckErrors("hipFree  fail");
    
//     hipDeviceReset();
    return 0;
}
void CreateTextureInterp(int num_devices,const float* imagedata,Geometry geo,hipArray** d_cuArrTex, hipTextureObject_t *texImage,bool allocate)
{
    //size_t size_image=geo.nVoxelX*geo.nVoxelY*geo.nVoxelZ;
    const hipExtent extent = make_hipExtent(geo.nVoxelX, geo.nVoxelY, geo.nVoxelZ);
    if(allocate){
        
        for (unsigned int i = 0; i < num_devices; i++){
            hipSetDevice(i);
            
            //hipArray Descriptor
            
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
            //cuda Array
            hipMalloc3DArray(&d_cuArrTex[i], &channelDesc, extent);
            cudaCheckErrors("Texture memory allocation fail");
        }
        
    }
    for (unsigned int i = 0; i < num_devices; i++){
        hipMemcpy3DParms copyParams = {0};
        hipSetDevice(i);
        //Array creation
        copyParams.srcPtr   = make_hipPitchedPtr((void *)imagedata, extent.width*sizeof(float), extent.width, extent.height);
        copyParams.dstArray = d_cuArrTex[i];
        copyParams.extent   = extent;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3DAsync(&copyParams);
        //cudaCheckErrors("Texture memory data copy fail");
        //Array creation End
    }
    for (unsigned int i = 0; i < num_devices; i++){
        hipSetDevice(i);
        hipResourceDesc    texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array  = d_cuArrTex[i];
        hipTextureDesc     texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        if (geo.accuracy>1){
            texDescr.filterMode = hipFilterModePoint;
            geo.accuracy=1;
        }
        else{
            texDescr.filterMode = hipFilterModeLinear;
        }
        texDescr.addressMode[0] = hipAddressModeBorder;
        texDescr.addressMode[1] = hipAddressModeBorder;
        texDescr.addressMode[2] = hipAddressModeBorder;
        texDescr.readMode = hipReadModeElementType;
        hipCreateTextureObject(&texImage[i], &texRes, &texDescr, NULL);
        cudaCheckErrors("Texture object creation fail");
    }
}

/* This code generates the geometries needed to split the image properly in
 * cases where the entire image does not fit in the memory of the GPU
 **/
void splitImageInterp(unsigned int splits,Geometry geo,Geometry* geoArray, unsigned int nangles){
    
    unsigned long splitsize=(geo.nVoxelZ+splits-1)/splits;// ceil if not divisible
    for(unsigned int sp=0;sp<splits;sp++){
        geoArray[sp]=geo;
        // All of them are splitsize, but the last one, possible
        geoArray[sp].nVoxelZ=((sp+1)*splitsize<geo.nVoxelZ)?  splitsize:  geo.nVoxelZ-splitsize*sp;
        geoArray[sp].sVoxelZ= geoArray[sp].nVoxelZ* geoArray[sp].dVoxelZ;
        
        // We need to redefine the offsets, as now each subimage is not aligned in the origin.
        geoArray[sp].offOrigZ=(float *)malloc(nangles*sizeof(float));
        for (unsigned int i=0;i<nangles;i++){
            geoArray[sp].offOrigZ[i]=geo.offOrigZ[i]-geo.sVoxelZ/2+sp*geoArray[0].sVoxelZ+geoArray[sp].sVoxelZ/2;
        }
        
    }
}



/* This code precomputes The location of the source and the Delta U and delta V (in the warped space)
 * to compute the locations of the x-rays. While it seems verbose and overly-optimized,
 * it does saves about 30% of each of the kernel calls. Thats something!
 **/
void computeDeltas(Geometry geo,unsigned int i, Point3D* uvorigin, Point3D* deltaU, Point3D* deltaV, Point3D* source){
    Point3D S;
    S.x=geo.DSO[i];
    S.y=0;
    S.z=0;
    
    //End point
    Point3D P,Pu0,Pv0;
    
    P.x  =-(geo.DSD[i]-geo.DSO[i]);   P.y  = geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       P.z  = geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pu0.x=-(geo.DSD[i]-geo.DSO[i]);   Pu0.y= geo.dDetecU*(1-((float)geo.nDetecU/2)+0.5);       Pu0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pv0.x=-(geo.DSD[i]-geo.DSO[i]);   Pv0.y= geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       Pv0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-1);
    // Geomtric trasnformations:
    
    
    // Now we have the Real world (OXYZ) coordinates of the bottom corner and its two neighbours.
    // The obkjective is to get a position of the detector in a coordinate system where:
    // 1-units are voxel size (in each direction can be different)
    // 2-The image has the its first voxel at (0,0,0)
    // 3-The image never rotates
    
    // To do that, we need to compute the "deltas" the detector, or "by how much
    // (in new xyz) does the voxels change when and index is added". To do that
    // several geometric steps needs to be changed
    
    //1.Roll,pitch,jaw
    // The detector can have a small rotation.
    // according to
    //"A geometric calibration method for cone beam CT systems" Yang K1, Kwan AL, Miller DF, Boone JM. Med Phys. 2006 Jun;33(6):1695-706.
    // Only the Z rotation will have a big influence in the image quality when they are small.
    // Still all rotations are supported
    
    // To roll pitch jaw, the detector has to be in centered in OXYZ.
    P.x=0;Pu0.x=0;Pv0.x=0;
    
    // Roll pitch yaw
    rollPitchYaw(geo,i,&P);
    rollPitchYaw(geo,i,&Pu0);
    rollPitchYaw(geo,i,&Pv0);
    //Now ltes translate the detector coordinates to DOD (original position on real coordinate system:
    P.x=P.x-(geo.DSD[i]-geo.DSO[i]);
    Pu0.x=Pu0.x-(geo.DSD[i]-geo.DSO[i]);
    Pv0.x=Pv0.x-(geo.DSD[i]-geo.DSO[i]);
    //2: Offset detector
    
    
    //S doesnt need to chagne
    
    
    
    //3: Rotate around RZ RY RZ
    Point3D Pfinal, Pfinalu0, Pfinalv0;
    Pfinal.x  =P.x;
    Pfinal.y  =P.y  +geo.offDetecU[i]; Pfinal.z  =P.z  +geo.offDetecV[i];
    Pfinalu0.x=Pu0.x;
    Pfinalu0.y=Pu0.y  +geo.offDetecU[i]; Pfinalu0.z  =Pu0.z  +geo.offDetecV[i];
    Pfinalv0.x=Pv0.x;
    Pfinalv0.y=Pv0.y  +geo.offDetecU[i]; Pfinalv0.z  =Pv0.z  +geo.offDetecV[i];
    
    eulerZYZ(geo,&Pfinal);
    eulerZYZ(geo,&Pfinalu0);
    eulerZYZ(geo,&Pfinalv0);
    eulerZYZ(geo,&S);
    
    
    //3: Offset image (instead of offseting image, -offset everything else)
    
    Pfinal.x  =Pfinal.x-geo.offOrigX[i];     Pfinal.y  =Pfinal.y-geo.offOrigY[i];     Pfinal.z  =Pfinal.z-geo.offOrigZ[i];
    Pfinalu0.x=Pfinalu0.x-geo.offOrigX[i];   Pfinalu0.y=Pfinalu0.y-geo.offOrigY[i];   Pfinalu0.z=Pfinalu0.z-geo.offOrigZ[i];
    Pfinalv0.x=Pfinalv0.x-geo.offOrigX[i];   Pfinalv0.y=Pfinalv0.y-geo.offOrigY[i];   Pfinalv0.z=Pfinalv0.z-geo.offOrigZ[i];
    S.x=S.x-geo.offOrigX[i];                 S.y=S.y-geo.offOrigY[i];                 S.z=S.z-geo.offOrigZ[i];
    
    // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
    Pfinal.x  =Pfinal.x+geo.sVoxelX/2-geo.dVoxelX/2;      Pfinal.y  =Pfinal.y+geo.sVoxelY/2-geo.dVoxelY/2;          Pfinal.z  =Pfinal.z  +geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalu0.x=Pfinalu0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalu0.y=Pfinalu0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalu0.z=Pfinalu0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalv0.x=Pfinalv0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalv0.y=Pfinalv0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalv0.z=Pfinalv0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    S.x       =S.x+geo.sVoxelX/2-geo.dVoxelX/2;           S.y       =S.y+geo.sVoxelY/2-geo.dVoxelY/2;               S.z       =S.z      +geo.sVoxelZ/2-geo.dVoxelZ/2;
    
    //4. Scale everything so dVoxel==1
    Pfinal.x  =Pfinal.x/geo.dVoxelX;      Pfinal.y  =Pfinal.y/geo.dVoxelY;        Pfinal.z  =Pfinal.z/geo.dVoxelZ;
    Pfinalu0.x=Pfinalu0.x/geo.dVoxelX;    Pfinalu0.y=Pfinalu0.y/geo.dVoxelY;      Pfinalu0.z=Pfinalu0.z/geo.dVoxelZ;
    Pfinalv0.x=Pfinalv0.x/geo.dVoxelX;    Pfinalv0.y=Pfinalv0.y/geo.dVoxelY;      Pfinalv0.z=Pfinalv0.z/geo.dVoxelZ;
    S.x       =S.x/geo.dVoxelX;           S.y       =S.y/geo.dVoxelY;             S.z       =S.z/geo.dVoxelZ;
    
    
    //mexPrintf("COR: %f \n",geo.COR[i]);
    //5. apply COR. Wherever everything was, now its offesetd by a bit.
//     Only wors for standard rotaiton, not aribtary axis rotation.
    float CORx, CORy;
    CORx=-geo.COR[i]*sin(geo.alpha)/geo.dVoxelX;
    CORy= geo.COR[i]*cos(geo.alpha)/geo.dVoxelY;
    Pfinal.x+=CORx;   Pfinal.y+=CORy;
    Pfinalu0.x+=CORx;   Pfinalu0.y+=CORy;
    Pfinalv0.x+=CORx;   Pfinalv0.y+=CORy;
    S.x+=CORx; S.y+=CORy;
    
    // return
    
    *uvorigin=Pfinal;
    
    deltaU->x=Pfinalu0.x-Pfinal.x;
    deltaU->y=Pfinalu0.y-Pfinal.y;
    deltaU->z=Pfinalu0.z-Pfinal.z;
    
    deltaV->x=Pfinalv0.x-Pfinal.x;
    deltaV->y=Pfinalv0.y-Pfinal.y;
    deltaV->z=Pfinalv0.z-Pfinal.z;
    
    *source=S;
}

float maxdistanceCuboid(Geometry geo,unsigned int i){
    ///////////
    // Compute initial "t" so we access safely as less as out of bounds as possible.
    //////////
    
    
    float maxCubX,maxCubY,maxCubZ;
    // Forgetting Z, compute mas distance: diagonal+offset
    maxCubX=(geo.nVoxelX/2+ abs(geo.offOrigX[i])/geo.dVoxelX);
    maxCubY=(geo.nVoxelY/2+ abs(geo.offOrigY[i])/geo.dVoxelY);
    maxCubZ=(geo.nVoxelZ/2+ abs(geo.offOrigZ[i])/geo.dVoxelZ);
    
    float a,b;
    a=geo.DSO[i]/geo.dVoxelX;
    b=geo.DSO[i]/geo.dVoxelY;
    
//  As the return of this value is in "voxel space", the source may have an elliptical curve.
//  The distance returned is the safe distance that can be skipped for a given angle alpha, before we need to start sampling.
    
    if (geo.theta==0.0f & geo.psi==0.0f) // Special case, it will make the code faster
        return max(a*b/sqrt(a*a*sin(geo.alpha)*sin(geo.alpha)+b*b*cos(geo.alpha)*cos(geo.alpha))-
                sqrt(maxCubX*maxCubX+maxCubY*maxCubY),0.0f);
    //TODO: think of more special cases?
    return max(geo.DSO[i]/max(max(geo.dVoxelX,geo.dVoxelY),geo.dVoxelZ)-sqrt(maxCubX*maxCubX+maxCubY*maxCubY+maxCubZ*maxCubZ),0.0f);
    
}
void rollPitchYaw(Geometry geo,unsigned int i, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=cos(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) - sin(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) + sin(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->y=sin(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) + cos(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) - cos(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->z=-sin(geo.dPitch[i])*auxPoint.x
            +cos(geo.dPitch[i])*sin(geo.dYaw[i])*auxPoint.y
            +cos(geo.dPitch[i])*cos(geo.dYaw[i])*auxPoint.z;
    
}
void eulerZYZ(Geometry geo,  Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=(+cos(geo.alpha)*cos(geo.theta)*cos(geo.psi)-sin(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-cos(geo.alpha)*cos(geo.theta)*sin(geo.psi)-sin(geo.alpha)*cos(geo.psi))*auxPoint.y+
            cos(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->y=(+sin(geo.alpha)*cos(geo.theta)*cos(geo.psi)+cos(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-sin(geo.alpha)*cos(geo.theta)*sin(geo.psi)+cos(geo.alpha)*cos(geo.psi))*auxPoint.y+
            sin(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->z=-sin(geo.theta)*cos(geo.psi)*auxPoint.x+
            sin(geo.theta)*sin(geo.psi)*auxPoint.y+
            cos(geo.theta)*auxPoint.z;
    
    
}
//______________________________________________________________________________
//
//      Function:       freeGeoArray
//
//      Description:    Frees the memory from the geometry array for multiGPU.
//______________________________________________________________________________
void freeGeoArray(unsigned int splits,Geometry* geoArray){
    for(unsigned int sp=0;sp<splits;sp++){
        free(geoArray[sp].offOrigZ);
    }
    free(geoArray);
}
//______________________________________________________________________________
//
//      Function:       checkFreeMemory
//
//      Description:    check available memory on devices
//______________________________________________________________________________
void checkFreeMemory(int deviceCount,size_t *mem_GPU_global){
    size_t memfree;
    size_t memtotal;
    
    for (int dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipMemGetInfo(&memfree,&memtotal);
        if(dev==0) *mem_GPU_global=memfree;
        if(memfree<memtotal/2){
            mexErrMsgIdAndTxt("tvDenoise:tvdenoising:GPU","One (or more) of your GPUs is being heavily used by another program (possibly graphics-based).\n Free the GPU to run TIGRE\n");
        }
        cudaCheckErrors("Check mem error");
        *mem_GPU_global=(memfree<*mem_GPU_global)?memfree:*mem_GPU_global;
    }
    *mem_GPU_global=(size_t)((double)*mem_GPU_global*0.95);
    
    //*mem_GPU_global= insert your known number here, in bytes.
}