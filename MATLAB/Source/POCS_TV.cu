#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for Steepest descend in POCS-type algorithms.
 *
 * This file will iteratively minimize by stepest descend the total variation
 * of the input image, with the parameters given, using GPUs.
 *
 * CODE by       Ander Biguri
 *
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */







#define MAXTHREADS 1024
#define MAX_BUFFER 60

#include "POCS_TV.hpp"




#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                hipDeviceReset();\
                mexErrMsgIdAndTxt("POCS_TV:GPU",hipGetErrorString(__err));\
        } \
} while (0)
    
// CUDA kernels
//https://stackoverflow.com/questions/21332040/simple-cuda-kernel-optimization/21340927#21340927
    __global__ void divideArrayScalar(float* vec,float scalar,const size_t n){
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]/=scalar;
        }
    }
    __global__ void multiplyArrayScalar(float* vec,float scalar,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]*=scalar;
        }
    }
    __global__ void substractArrays(float* vec,float* vec2,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]-=vec2[i];
        }
    }
    
    __device__ __inline__
            void gradient(const float* u, float* grad,
            long z, long y, long x,
            long depth, long rows, long cols){
        unsigned long size2d = rows*cols;
        unsigned long long idx = z * size2d + y * cols + x;
        
        float uidx = u[idx];
        
        if ( z - 1 >= 0 && z<depth) {
            grad[0] = (uidx-u[(z-1)*size2d + y*cols + x]) ;
        }
        
        if ( y - 1 >= 0 && y<rows){
            grad[1] = (uidx-u[z*size2d + (y-1)*cols + x]) ;
        }
        
        if ( x - 1 >= 0 && x<cols) {
            grad[2] = (uidx-u[z*size2d + y*cols + (x-1)]);
        }
    }
    
    __global__ void gradientTV(const float* f, float* dftv,
            long depth, long rows, long cols){
        unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
        unsigned long z = threadIdx.z + blockIdx.z * blockDim.z;
        unsigned long long idx = z * rows * cols + y * cols + x;
        if ( x >= cols || y >= rows || z >= depth )
            return;
        
        
        float df[3] ={0.f,0.f,0.f};
        float dfi[3]={0.f,0.f,0.f}; // dfi== \partial f_{i+1,j,k}
        float dfj[3]={0.f,0.f,0.f};
        float dfk[3]={0.f,0.f,0.f};
        gradient(f,df  ,z  ,y  ,x  , depth,rows,cols);
        gradient(f,dfi ,z  ,y  ,x+1, depth,rows,cols);
        gradient(f,dfj ,z  ,y+1,x  , depth,rows,cols);
        gradient(f,dfk ,z+1,y  ,x  , depth,rows,cols);
        float eps=0.00000001; //% avoid division by zero
        
        dftv[idx]=(df[0]+df[1]+df[2])/(sqrt(df[0] *df[0] +df[1] *df[1] +df[2] *df[2])+eps)
        -dfi[2]/(sqrt(dfi[0]*dfi[0]+dfi[1]*dfi[1]+dfi[2]*dfi[2]) +eps)     // I wish I coudl precompute this, but if I do then Id need to recompute the gradient.
        -dfj[1]/(sqrt(dfj[0]*dfj[0]+dfj[1]*dfj[1]+dfj[2]*dfj[2]) +eps)
        -dfk[0]/(sqrt(dfk[0]*dfk[0]+dfk[1]*dfk[1]+dfk[2]*dfk[2]) +eps);
        return;
        
    }
    
    __device__ void warpReduce(volatile float *sdata, size_t tid) {
        sdata[tid] += sdata[tid + 32];
        sdata[tid] += sdata[tid + 16];
        sdata[tid] += sdata[tid + 8];
        sdata[tid] += sdata[tid + 4];
        sdata[tid] += sdata[tid + 2];
        sdata[tid] += sdata[tid + 1];
    }
    
    __global__ void  reduceNorm2(float *g_idata, float *g_odata, size_t n){
        extern __shared__ volatile float sdata[];
        //http://stackoverflow.com/a/35133396/1485872
        size_t tid = threadIdx.x;
        size_t i = blockIdx.x*blockDim.x + tid;
        size_t gridSize = blockDim.x*gridDim.x;
        float mySum = 0;
        float value=0;
        while (i < n) {
            value=g_idata[i]; //avoid reading twice
            mySum += value*value;
            i += gridSize;
        }
        sdata[tid] = mySum;
        __syncthreads();
        
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
        
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
        
        if (tid <  64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
        
        
#if (__CUDA_ARCH__ >= 300)
        if ( tid < 32 )
        {
            mySum = sdata[tid] + sdata[tid + 32];
            for (int offset = warpSize/2; offset > 0; offset /= 2) {
                mySum += __shfl_down_sync(0xFFFFFFFF, mySum, offset,32);
            }
        }
#else
        if (tid < 32) {
            warpReduce(sdata, tid);
            mySum = sdata[0];
        }
#endif
        if (tid == 0) g_odata[blockIdx.x] = mySum;
    }
    
    __global__ void  reduceSum(float *g_idata, float *g_odata, size_t n){
        extern __shared__ volatile float sdata[];
        //http://stackoverflow.com/a/35133396/1485872
        size_t tid = threadIdx.x;
        size_t i = blockIdx.x*blockDim.x + tid;
        size_t gridSize = blockDim.x*gridDim.x;
        float mySum = 0;
        // float value=0;
        while (i < n) {
            mySum += g_idata[i];
            i += gridSize;
        }
        sdata[tid] = mySum;
        __syncthreads();
        
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
        
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
        
        if (tid <  64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
        
        
#if (__CUDA_ARCH__ >= 300)
        if ( tid < 32 )
        {
            mySum = sdata[tid] + sdata[tid + 32];
            for (int offset = warpSize/2; offset > 0; offset /= 2) {
                mySum += __shfl_down_sync(0xFFFFFFFF, mySum, offset,32);
            }
        }
#else
        if (tid < 32) {
            warpReduce(sdata, tid);
            mySum = sdata[0];
        }
#endif
        if (tid == 0) g_odata[blockIdx.x] = mySum;
    }
    
    
    
    
// main function
    void pocs_tv(float* img,float* dst,float alpha,const long* image_size, int maxIter){
        
        
       
        
        // Prepare for MultiGPU
        int deviceCount = 0;
        hipGetDeviceCount(&deviceCount);
        cudaCheckErrors("Device query fail");
        if (deviceCount == 0) {
            mexErrMsgIdAndTxt("POCS_TV:GPU","There are no available device(s) that support CUDA\n");
        }
        //
        // CODE assumes
        // 1.-All available devices are usable by this code
        // 2.-All available devices are equal, they are the same machine (warning trhown)
        int dev;
        char * devicenames;
        hipDeviceProp_t deviceProp;
        
        for (dev = 0; dev < deviceCount; dev++) {
            hipSetDevice(dev);
            hipGetDeviceProperties(&deviceProp, dev);
            if (dev>0){
                if (strcmp(devicenames,deviceProp.name)!=0){
                    mexWarnMsgIdAndTxt("minimizeTV:POCS_TV:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed. \n POCS_TV.cu line 277.");
                    break;
                }
            }
            devicenames=deviceProp.name;
        }
        
        
        // We don't know if the devices are being used. lets check that. and only use the amount of memory we need.

        size_t mem_GPU_global;
        checkFreeMemory(deviceCount,&mem_GPU_global);

        
        
        // %5 of free memory shoudl be enough, we have almsot no variables in these kernels
        size_t total_pixels              = image_size[0] * image_size[1]  * image_size[2] ;
        size_t mem_slice_image           = sizeof(float)* image_size[0] * image_size[1]  ;
        size_t mem_size_image            = sizeof(float)* total_pixels;
        size_t mem_auxiliary             = sizeof(float)* (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
        
        // Decide how are we handling the distribution of computation
        size_t mem_img_each_GPU;
        
        unsigned int buffer_length=2;
        //Does everything fit in the GPU?
        unsigned int slices_per_split;
        unsigned int splits=1; // if the number does not fit in an uint, you have more serious trouble than this.
        if(mem_GPU_global> 3*mem_size_image+3*(deviceCount-1)*mem_slice_image*buffer_length+mem_auxiliary){
            // We only need to split if we have extra GPUs
            slices_per_split=(image_size[2]+deviceCount-1)/deviceCount;
            mem_img_each_GPU=mem_slice_image*((slices_per_split+buffer_length*2));
        }else{
            // As mem_auxiliary is not expected to be a large value (for a 2000^3 image is around 28Mbytes), lets for now assume we need it all
            size_t mem_free=mem_GPU_global-mem_auxiliary;
            
            splits=(unsigned int)(ceil(((float)(3*mem_size_image)/(float)(deviceCount))/mem_free));
            // Now, there is an overhead here, as each splits should have 2 slices more, to accoutn for overlap of images.
            // lets make sure these 2 slices fit, if they do not, add 1 to splits.
            slices_per_split=(image_size[2]+deviceCount*splits-1)/(deviceCount*splits);
            mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));
            
            // if the new stuff does not fit in the GPU, it measn we are in the edge case where adding that extra slice will overflow memory
            if (mem_GPU_global< 3*mem_img_each_GPU+mem_auxiliary){
                // one more splot shoudl do the job, as its an edge case.
                splits++;
                //recompute for later
                slices_per_split=(image_size[2]+deviceCount*splits-1)/(deviceCount*splits); // amountf of slices that fit on a GPU. Later we add 2 to these, as we need them for overlap
                mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));
            }


            // How many EXTRA buffer slices should be able to fit in here??!?!
            // Only do it if there are splits needed. 
            if(splits>1){
                mem_free=mem_GPU_global-(3*mem_img_each_GPU+mem_auxiliary);
                unsigned int extra_buff=(mem_free/mem_slice_image); 
                buffer_length=(extra_buff/2)/3; // we need double whatever this results in, rounded down.
                buffer_length=max(buffer_length,2);// minimum 2
                buffer_length=min(MAX_BUFFER,buffer_length);

                mem_img_each_GPU=mem_slice_image*(slices_per_split+buffer_length*2);
                
            }else{
                buffer_length=2;
            }

            // Assert
            if (mem_GPU_global< 3*mem_img_each_GPU+mem_auxiliary){
                mexErrMsgIdAndTxt("POCS_TV:GPU","Assertion Failed. Logic behind spliting flawed! Please tell: ander.biguri@gmail.com\n");
            }
        }
        
        
         // Assert
       
        if ((slices_per_split+buffer_length*2)*image_size[0]*image_size[1]* sizeof(float)!= mem_img_each_GPU){
            mexErrMsgIdAndTxt("POCS_TV:GPU","Assertion Failed. Memory needed calculation broken! Please tell: ander.biguri@gmail.com\n");
        }
        
        
        
        
        
        
        float** d_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** d_dimgTV=   (float**)malloc(deviceCount*sizeof(float*));
        float** d_norm2aux= (float**)malloc(deviceCount*sizeof(float*));
        float** d_norm2=    (float**)malloc(deviceCount*sizeof(float*));
         
        // allocate memory in each GPU
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            
            hipMalloc((void**)&d_image[dev]    , mem_img_each_GPU);
            hipMemset(d_image[dev],0           , mem_img_each_GPU);
            hipMalloc((void**)&d_dimgTV[dev]   , mem_img_each_GPU);
            hipMalloc((void**)&d_norm2[dev]    , slices_per_split*mem_slice_image);
            hipMalloc((void**)&d_norm2aux[dev] , mem_auxiliary);
            cudaCheckErrors("Malloc  error");
            
            
        }
       unsigned long long buffer_pixels=buffer_length*image_size[0]*image_size[1];
        float* buffer;
        if(splits>1){
            mexWarnMsgIdAndTxt("minimizeTV:POCS_TV:Image_split","Your image can not be fully split between the available GPUs. The computation of minTV will be significantly slowed due to the image size.\nApproximated mathematics turned on for computational speed.");
        }else{
            hipHostMalloc((void**)&buffer,buffer_length*image_size[0]*image_size[1]*sizeof(float));
        }
        
        
        
        // Lets try to make the host memory pinned:
        // We laredy queried the GPU and assuemd they are the same, thus shoudl have the same attributes.
        int isHostRegisterSupported;
        hipDeviceGetAttribute(&isHostRegisterSupported,hipDeviceAttributeHostRegisterSupported,0);
        // splits>2 is completely empirical observation
        if (isHostRegisterSupported & splits>2){
            hipHostRegister(img ,image_size[2]*image_size[1]*image_size[0]*sizeof(float),hipHostRegisterPortable);
            hipHostRegister(dst ,image_size[2]*image_size[1]*image_size[0]*sizeof(float),hipHostRegisterPortable);
        }
        cudaCheckErrors("Error pinning memory");

        
        
                // Create streams
        int nStream_device=2;
        int nStreams=deviceCount*nStream_device;
        hipStream_t* stream=(hipStream_t*)malloc(nStreams*sizeof(hipStream_t));
        
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            for (int i = 0; i < nStream_device; ++i){
                hipStreamCreate(&stream[i+dev*nStream_device]);
            }
        }
        cudaCheckErrors("Stream creation fail");

        
        // For the reduction

        double totalsum_prev;
        double totalsum;
        float sum_curr_spl;
        float * sumnorm2;
        hipHostMalloc((void**)&sumnorm2,deviceCount*sizeof(float));
        
        unsigned int curr_slices;
        unsigned long long curr_pixels;
        size_t linear_idx_start;
        unsigned long long* offset_device=(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        unsigned long long* offset_host  =(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        unsigned long long* bytes_device =(unsigned long long*)malloc(deviceCount*sizeof(unsigned long long));
        bool is_first_chunk;
        bool is_last_chunk;
        for(unsigned int i=0;i<maxIter;i+=(buffer_length-1)){
            if(splits>1){
                totalsum_prev=0;
            }
            for(unsigned int sp=0;sp<splits;sp++){
                
                // For each iteration we need to comptue all the image. The ordering of these loops
                // need to be like this due to the boudnign layers between slpits. If more than 1 split is needed
                // for each GPU then there is no other way that taking the entire memory out of GPU and putting it back.
                // If the memory can be shared ebtween GPUs fully without extra splits, then there is an easy way of syncronizing the memory
                
                // Copy image to memory
                for (dev = 0; dev < deviceCount; dev++){
                    curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                    curr_pixels=curr_slices*image_size[0]*image_size[1];
                    linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                    
                    // Check if its the first or last chunck
                    is_last_chunk=!((sp*deviceCount+dev)<deviceCount*splits-1);
                    is_first_chunk=!(sp*deviceCount+dev);
                    
                    // lets compute where we start copyes and how much. This avoids 3 calls to Memcpy
                    offset_device[dev]=buffer_pixels*is_first_chunk;
                    offset_host[dev]=linear_idx_start-buffer_pixels*!is_first_chunk;
                    bytes_device[dev]=curr_pixels+buffer_pixels*!is_first_chunk+buffer_pixels*!is_last_chunk;
                }

                if(i==0){
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        
                        hipMemcpyAsync(d_image[dev]+offset_device[dev], img+offset_host[dev]  , bytes_device[dev]*sizeof(float), hipMemcpyHostToDevice,stream[dev*nStream_device+1]);
                        
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        hipDeviceSynchronize();
                    }
                }
                // if we need to split and its not the first iteration, then we need to copy from Host memory the previosu result.
                if (splits>1 & i>0){
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        hipMemcpyAsync(d_image[dev]+offset_device[dev], dst+offset_host[dev]  , bytes_device[dev]*sizeof(float), hipMemcpyHostToDevice,stream[dev*nStream_device+1]);
                        
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        hipDeviceSynchronize();
                    }
                }
                cudaCheckErrors("Memcpy failure on multi split");
                
                for(unsigned int ib=0;  (ib<(buffer_length-1)) && ((i+ib)<maxIter);  ib++){
                    
                    // For the gradient
                    dim3 blockGrad(10, 10, 10);
                    dim3 gridGrad((image_size[0]+blockGrad.x-1)/blockGrad.x, (image_size[1]+blockGrad.y-1)/blockGrad.y, (curr_slices+buffer_length*2+blockGrad.z-1)/blockGrad.z);
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        // Compute the gradient of the TV norm
                        
                        // I Dont understand why I need to store 2 layers to compute correctly with 1 buffer. The bounding checks shoudl
                        // be enough but they are not.
                        gradientTV<<<gridGrad, blockGrad,0,stream[dev*nStream_device]>>>(d_image[dev],d_dimgTV[dev],(long)(curr_slices+buffer_length*2-1), image_size[1],image_size[0]);
                        
                    }
                    
                    
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        // no need to copy the 2 aux slices here
                        hipStreamSynchronize(stream[dev*nStream_device]);
                        hipMemcpyAsync(d_norm2[dev], d_dimgTV[dev]+buffer_pixels, image_size[0]*image_size[1]*curr_slices*sizeof(float), hipMemcpyDeviceToDevice,stream[dev*nStream_device+1]);
                    }
                    
                    
                    // Compute the L2 norm of the gradint. For that, reduction is used.
                    //REDUCE
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        size_t dimblockRed = MAXTHREADS;
                        size_t dimgridRed = (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
                        
                        hipStreamSynchronize(stream[dev*nStream_device+1]);
                        reduceNorm2 << <dimgridRed, dimblockRed, MAXTHREADS*sizeof(float),stream[dev*nStream_device]>> >(d_norm2[dev], d_norm2aux[dev], total_pixels);
                        
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        size_t dimblockRed = MAXTHREADS;
                        size_t dimgridRed = (total_pixels + MAXTHREADS - 1) / MAXTHREADS;

                        if (dimgridRed > 1) {
                            reduceSum << <1, dimblockRed, MAXTHREADS*sizeof(float),stream[dev*nStream_device] >> >(d_norm2aux[dev], d_norm2[dev], dimgridRed);
                            hipStreamSynchronize(stream[dev*nStream_device]);
                            hipMemcpyAsync(&sumnorm2[dev], d_norm2[dev], sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                        }
                        else {
                            hipStreamSynchronize(stream[dev*nStream_device]);
                            hipMemcpyAsync(&sumnorm2[dev], d_norm2aux[dev], sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                        }
                    }
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        hipDeviceSynchronize();
                     }
                    cudaCheckErrors("Reduction error");
                    
                    
                    // Accumulate the nomr accross devices
                    sum_curr_spl=0;
                    // this is CPU code
                    for (dev = 0; dev < deviceCount; dev++){
                        sum_curr_spl+=sumnorm2[dev];
                    }
                    sum_curr_spl+=0.0000001f; // avoid division by zero
                    
                    // If we have more than one splits, lets use the result from prior calls
                    if(i>0 && splits>1){
                        // this is already stored:
                        //totalsum=totalsum_prev; 
                    }else{
                        totalsum=sum_curr_spl;
                    }
                    
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        //NOMRALIZE
                        //in a Tesla, maximum blocks =15 SM * 4 blocks/SM
                        divideArrayScalar  <<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_dimgTV[dev]+buffer_pixels,(float)sqrt(totalsum),total_pixels);
                        //MULTIPLY HYPERPARAMETER
                        multiplyArrayScalar<<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_dimgTV[dev]+buffer_pixels,alpha,   total_pixels);
                    }
                     for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        hipDeviceSynchronize();
                     }
                    cudaCheckErrors("Scalar operations error");
                    
                    //SUBSTRACT GRADIENT
                    //////////////////////////////////////////////
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        substractArrays<<<60,MAXTHREADS,0,stream[dev*nStream_device]>>>(d_image[dev]+buffer_pixels,d_dimgTV[dev]+buffer_pixels, total_pixels);
                    }
                }

                // Syncronize mathematics, make sure bounding pixels are correct
                 for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        hipDeviceSynchronize();
                     }
                
                if(splits==1){
                    for(dev=0; dev<deviceCount;dev++){
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        if (dev<deviceCount-1){
                            hipSetDevice(dev+1);
                            hipMemcpy(buffer, d_image[dev+1], buffer_pixels*sizeof(float), hipMemcpyDeviceToHost);
                            hipSetDevice(dev);
                            hipMemcpy(d_image[dev]+total_pixels+buffer_pixels,buffer, buffer_pixels*sizeof(float), hipMemcpyHostToDevice); 
                        }
                        hipDeviceSynchronize();
                        if (dev>0){
                            hipSetDevice(dev-1);
                            hipMemcpyAsync(buffer, d_image[dev-1]+total_pixels+buffer_pixels, buffer_pixels*sizeof(float), hipMemcpyDeviceToHost);
                            hipSetDevice(dev);
                            hipMemcpyAsync(d_image[dev],buffer, buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                        }
                    }
                }else{
                    
                    // We need to take it out :(
                    for(dev=0; dev<deviceCount;dev++){
                        hipSetDevice(dev);
                        
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        hipMemcpyAsync(&dst[linear_idx_start], d_image[dev]+buffer_pixels,total_pixels*sizeof(float), hipMemcpyDeviceToHost,stream[dev*nStream_device+1]);
                    }
                }
                
                for (dev = 0; dev < deviceCount; dev++){
                    hipSetDevice(dev);
                    hipDeviceSynchronize();
                }
                cudaCheckErrors("Memory gather error");

                totalsum_prev+=sum_curr_spl;
            }
            totalsum=totalsum_prev;
        }
        // If there has not been splits, we still have data in memory
        if(splits==1){
            for(dev=0; dev<deviceCount;dev++){
                hipSetDevice(dev);
                
                curr_slices=((dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*dev;
                total_pixels=curr_slices*image_size[0]*image_size[1];
                hipMemcpy(dst+slices_per_split*image_size[0]*image_size[1]*dev, d_image[dev]+buffer_pixels,total_pixels*sizeof(float), hipMemcpyDeviceToHost);
            }
        }
        cudaCheckErrors("Copy result back");
        
        for(dev=0; dev<deviceCount;dev++){
            hipSetDevice(dev);
            hipFree(d_image[dev]);
            hipFree(d_norm2aux[dev]);
            hipFree(d_dimgTV[dev]);
            hipFree(d_norm2[dev]);
        }
        if (splits==1){
            hipHostFree(buffer);
        }
        
        if (isHostRegisterSupported& splits>2){
            hipHostUnregister(img);
            hipHostUnregister(dst);
        }
        for (int i = 0; i < nStreams; ++i)
           hipStreamDestroy(stream[i]) ;
        
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipDeviceSynchronize();
        }
        cudaCheckErrors("Memory free");
        hipDeviceReset();
    }
        
void checkFreeMemory(int deviceCount,size_t *mem_GPU_global){
        size_t memfree;
        size_t memtotal;
        
        for (int dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipMemGetInfo(&memfree,&memtotal);
            if(dev==0) *mem_GPU_global=memfree;
            if(memfree<memtotal/2){
                mexErrMsgIdAndTxt("POCS_TV:GPU","One (or more) of your GPUs is being heavily used by another program (possibly graphics-based).\n Free the GPU to run TIGRE\n");
            }
            cudaCheckErrors("Check mem error");
            
            *mem_GPU_global=(memfree<*mem_GPU_global)?memfree:*mem_GPU_global;
        }
        *mem_GPU_global=(size_t)((double)*mem_GPU_global*0.95);
        
        //*mem_GPU_global= insert your known number here, in bytes.
}
