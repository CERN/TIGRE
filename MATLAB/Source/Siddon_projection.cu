#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for ray-voxel intersection based projection
 *
 * This file has the necesary fucntiosn to perform X-ray CBCT projection
 * operation given a geaometry, angles and image. It usesthe so-called
 * Jacobs algorithm to compute efficiently the length of the x-rays over
 * voxel space.
 *
 * CODE by       Ander Biguri
 *               Sepideh Hatamikia (arbitrary rotation)
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "Siddon_projection.hpp"
#include "mex.h"
#include <math.h>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("Ax:Siddon_projection",hipGetErrorString(__err));\
        } \
} while (0)
    
    
#define MAXTREADS 1024
#define PROJ_PER_BLOCK 9
#define PIXEL_SIZE_BLOCK 9
    /*GEOMETRY DEFINITION
     *
     *                Detector plane, behind
     *            |-----------------------------|
     *            |                             |
     *            |                             |
     *            |                             |
     *            |                             |
     *            |      +--------+             |
     *            |     /        /|             |
     *   A Z      |    /        / |*D           |
     *   |        |   +--------+  |             |
     *   |        |   |        |  |             |
     *   |        |   |     *O |  +             |
     *    --->y   |   |        | /              |
     *  /         |   |        |/               |
     * V X        |   +--------+                |
     *            |-----------------------------|
     *
     *           *S
     *
     *
     *
     *
     *
     **/
    
    void CreateTexture(int num_devices,const float* imagedata,Geometry geo,hipArray** d_cuArrTex, hipTextureObject_t *texImage,bool alloc);

__constant__ Point3D projParamsArrayDev[4*PROJ_PER_BLOCK];  // Dev means it is on device


__global__ void vecAddInPlace(float *a, float *b, unsigned long  n)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (idx < n)
        a[idx] = a[idx] + b[idx];
}

__global__ void kernelPixelDetector( Geometry geo,
        float* detector,
        const int currProjSetNumber,
        const int totalNoOfProjections,
        hipTextureObject_t tex){
    
    
    unsigned long y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long projNumber=threadIdx.z;
    
    
    if ((x>= geo.nDetecU) | (y>= geo.nDetecV)|  (projNumber>=PROJ_PER_BLOCK))
        return;
    
    size_t idx =  (size_t)(x  * geo.nDetecV + y)+ (size_t)projNumber*geo.nDetecV *geo.nDetecU ;
    int indAlpha = currProjSetNumber*PROJ_PER_BLOCK+projNumber;  // This is the ABSOLUTE projection number in the projection array (for a given GPU)

    if(indAlpha>=totalNoOfProjections)
        return;
    
    Point3D uvOrigin = projParamsArrayDev[4*projNumber];  // 6*projNumber because we have 6 Point3D values per projection
    Point3D deltaU = projParamsArrayDev[4*projNumber+1];
    Point3D deltaV = projParamsArrayDev[4*projNumber+2];
    Point3D source = projParamsArrayDev[4*projNumber+3];
    
    /////// Get coordinates XYZ of pixel UV
    int pixelV = geo.nDetecV-y-1;
    int pixelU = x;
    Point3D pixel1D;
    pixel1D.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    pixel1D.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    pixel1D.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    ///////
    // Siddon's ray-voxel intersection, optimized as in doi=10.1.1.55.7516
    //////
    // Also called Jacobs algorithms
    Point3D ray;
    // vector of Xray
    ray.x=pixel1D.x-source.x;
    ray.y=pixel1D.y-source.y;
    ray.z=pixel1D.z-source.z;
    // This variables are ommited because
    // bx,by,bz ={0,0,0}
    // dx,dy,dz ={1,1,1}
    // compute parameter values for x-ray parametric equation. eq(3-10)
    float axm,aym,azm;
    float axM,ayM,azM;
    // In the paper Nx= number of X planes-> Nvoxel+1
   
    axm=fminf(__fdividef(-source.x,ray.x),__fdividef(geo.nVoxelX-source.x,ray.x));
    aym=fminf(__fdividef(-source.y,ray.y),__fdividef(geo.nVoxelY-source.y,ray.y));
    azm=fminf(__fdividef(-source.z,ray.z),__fdividef(geo.nVoxelZ-source.z,ray.z));
    axM=fmaxf(__fdividef(-source.x,ray.x),__fdividef(geo.nVoxelX-source.x,ray.x));
    ayM=fmaxf(__fdividef(-source.y,ray.y),__fdividef(geo.nVoxelY-source.y,ray.y));
    azM=fmaxf(__fdividef(-source.z,ray.z),__fdividef(geo.nVoxelZ-source.z,ray.z));
    
    float am=fmaxf(fmaxf(axm,aym),azm);
    float aM=fminf(fminf(axM,ayM),azM);
    
    // line intersects voxel space ->   am<aM
    if (am>=aM)
        detector[idx]=0;
    
    // Compute max/min image INDEX for intersection eq(11-19)
    // Discussion about ternary operator in CUDA: https://stackoverflow.com/questions/7104384/in-cuda-why-is-a-b010-more-efficient-than-an-if-else-version
    float imin,imax,jmin,jmax,kmin,kmax;
    // for X
    if( source.x<pixel1D.x){
        imin=(am==axm)? 1.0f             : ceilf (source.x+am*ray.x);
        imax=(aM==axM)? geo.nVoxelX      : floorf(source.x+aM*ray.x);
    }else{
        imax=(am==axm)? geo.nVoxelX-1.0f : floorf(source.x+am*ray.x);
        imin=(aM==axM)? 0.0f             : ceilf (source.x+aM*ray.x);
    }
    // for Y
    if( source.y<pixel1D.y){
        jmin=(am==aym)? 1.0f             : ceilf (source.y+am*ray.y);
        jmax=(aM==ayM)? geo.nVoxelY      : floorf(source.y+aM*ray.y);
    }else{
        jmax=(am==aym)? geo.nVoxelY-1.0f : floorf(source.y+am*ray.y);
        jmin=(aM==ayM)? 0.0f             : ceilf (source.y+aM*ray.y);
    }
    // for Z
    if( source.z<pixel1D.z){
        kmin=(am==azm)? 1.0f             : ceilf (source.z+am*ray.z);
        kmax=(aM==azM)? geo.nVoxelZ      : floorf(source.z+aM*ray.z);
    }else{
        kmax=(am==azm)? geo.nVoxelZ-1.0f : floorf(source.z+am*ray.z);
        kmin=(aM==azM)? 0.0f             : ceilf (source.z+aM*ray.z);
    }
    
    // get intersection point N1. eq(20-21) [(also eq 9-10)]
    float ax,ay,az;
    ax=(source.x<pixel1D.x)?  __fdividef(imin-source.x,ray.x+0.000000000001f) :  __fdividef(imax-source.x,ray.x+0.000000000001f);
    ay=(source.y<pixel1D.y)?  __fdividef(jmin-source.y,ray.y+0.000000000001f) :  __fdividef(jmax-source.y,ray.y+0.000000000001f);
    az=(source.z<pixel1D.z)?  __fdividef(kmin-source.z,ray.z+0.000000000001f) :  __fdividef(kmax-source.z,ray.z+0.000000000001f);
    
    
    
    // get index of first intersection. eq (26) and (19)
    int i,j,k;
    float aminc=fminf(fminf(ax,ay),az);
    i=(int)floorf(source.x+ (aminc+am)*0.5f*ray.x);
    j=(int)floorf(source.y+ (aminc+am)*0.5f*ray.y);
    k=(int)floorf(source.z+ (aminc+am)*0.5f*ray.z);
    // Initialize
    float ac=am;
    //eq (28), unit anlges
    float axu,ayu,azu;
    axu=__frcp_rd(fabsf(ray.x));
    ayu=__frcp_rd(fabsf(ray.y));
    azu=__frcp_rd(fabsf(ray.z));
    // eq(29), direction of update
    float iu,ju,ku;
    iu=(source.x< pixel1D.x)? 1.0f : -1.0f;
    ju=(source.y< pixel1D.y)? 1.0f : -1.0f;
    ku=(source.z< pixel1D.z)? 1.0f : -1.0f;
    
    float maxlength=__fsqrt_rd(ray.x*ray.x*geo.dVoxelX*geo.dVoxelX+ray.y*ray.y*geo.dVoxelY*geo.dVoxelY+ray.z*ray.z*geo.dVoxelZ*geo.dVoxelZ);
    float sum=0.0f;
    unsigned int Np=(imax-imin+1)+(jmax-jmin+1)+(kmax-kmin+1); // Number of intersections
    // Go iterating over the line, intersection by intersection. If double point, no worries, 0 will be computed
    i+=0.5f;
    j+=0.5f;
    k+=0.5f;
    for (unsigned int ii=0;ii<Np;ii++){
        if (ax==aminc){
            sum+=(ax-ac)*tex3D<float>(tex, i, j, k);
            i=i+iu;
            ac=ax;
            ax+=axu;
        }else if(ay==aminc){
            sum+=(ay-ac)*tex3D<float>(tex, i, j, k);
            j=j+ju;
            ac=ay;
            ay+=ayu;
        }else if(az==aminc){
            sum+=(az-ac)*tex3D<float>(tex, i, j, k);
            k=k+ku;
            ac=az;
            az+=azu;
        }
        aminc=fminf(fminf(ax,ay),az);
    }
    detector[idx]=sum*maxlength;
}


int siddon_ray_projection(float  *  img, Geometry geo, float** result,float const * const angles,int nangles){

    
    // Prepare for MultiGPU
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    cudaCheckErrors("Device query fail");
    if (deviceCount == 0) {
        mexErrMsgIdAndTxt("Ax:Siddon_projection:GPUselect","There are no available device(s) that support CUDA\n");
    }
    //
    // CODE assumes
    // 1.-All available devices are usable by this code
    // 2.-All available devices are equal, they are the same machine (warning thrown)
    int dev;
    char * devicenames;
    hipDeviceProp_t deviceProp;
    
    for (dev = 0; dev < deviceCount; dev++) {
        hipSetDevice(dev);
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev>0){
            if (strcmp(devicenames,deviceProp.name)!=0){
                mexWarnMsgIdAndTxt("Ax:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed. \n Siddon_projection.cu line 275.");
                break;
            }
        }
        devicenames=deviceProp.name;
    }
    
    
    
    
    // Check free memory
    size_t mem_GPU_global;
    checkFreeMemory(deviceCount,&mem_GPU_global);

    size_t mem_image=                 (unsigned long long)geo.nVoxelX*(unsigned long long)geo.nVoxelY*(unsigned long long)geo.nVoxelZ*sizeof(float);
    size_t mem_proj=                  (unsigned long long)geo.nDetecU*(unsigned long long)geo.nDetecV*sizeof(float);
    
    // Does everything fit in the GPUs?
    bool fits_in_memory=false;
    unsigned int splits=1;
    Geometry * geoArray;
    
    
    if (mem_image+2*PROJ_PER_BLOCK*mem_proj<mem_GPU_global){// yes it does
        fits_in_memory=true;
        geoArray=(Geometry*)malloc(sizeof(Geometry));
        geoArray[0]=geo;
    }
    else{// Nope nope.
        fits_in_memory=false; // Oh dear.
        // approx free memory we have. We already have left some extra 5% free for internal stuff
        // we need a second projection memory to combine multi-GPU stuff.
        size_t mem_free=mem_GPU_global-4*PROJ_PER_BLOCK*mem_proj;
        
        
        splits=mem_image/mem_free+1;// Ceil of the truncation
        geoArray=(Geometry*)malloc(splits*sizeof(Geometry));
        splitImage(splits,geo,geoArray,nangles);
    }
    
    // Allocate axuiliary memory for projections on the GPU to accumulate partial results
    float ** dProjection_accum;
    size_t num_bytes_proj = PROJ_PER_BLOCK*geo.nDetecU*geo.nDetecV * sizeof(float);
    if (!fits_in_memory){
        dProjection_accum=(float**)malloc(2*deviceCount*sizeof(float*));
        for (dev = 0; dev < deviceCount; dev++) {
            hipSetDevice(dev);
            for (int i = 0; i < 2; ++i){
                hipMalloc((void**)&dProjection_accum[dev*2+i], num_bytes_proj);
                hipMemset(dProjection_accum[dev*2+i],0,num_bytes_proj);
                cudaCheckErrors("cudaMallocauxiliarty projections fail");
            }
        }
    }
    
    // This is happening regarthless if the image fits on memory
    float** dProjection=(float**)malloc(2*deviceCount*sizeof(float*));
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        
        for (int i = 0; i < 2; ++i){
            hipMalloc((void**)&dProjection[dev*2+i],   num_bytes_proj);
            hipMemset(dProjection[dev*2+i]  ,0,num_bytes_proj);
            cudaCheckErrors("hipMalloc projections fail");
        }
    }
    
    
    //Pagelock memory for syncronous copy.
    // Lets try to make the host memory pinned:
    // We laredy queried the GPU and assuemd they are the same, thus shoudl have the same attributes.
    int isHostRegisterSupported;
    hipDeviceGetAttribute(&isHostRegisterSupported,hipDeviceAttributeHostRegisterSupported,0);
    // empirical testing shows that when the image split is smaller than 1 (also implies the image is not very big), the time to
    // pin the memory is greater than the lost time in Syncronously launching the memcpys. This is only worth it when the image is too big.
    if (isHostRegisterSupported & (splits>1 |deviceCount>1)){
        hipHostRegister(img, (size_t)geo.nVoxelX*(size_t)geo.nVoxelY*(size_t)geo.nVoxelZ*(size_t)sizeof(float),hipHostRegisterPortable);
    }

    
    
    // auxiliary variables
    Point3D source, deltaU, deltaV, uvOrigin;
    Point3D* projParamsArrayHost;
    hipHostMalloc((void**)&projParamsArrayHost,4*PROJ_PER_BLOCK*sizeof(Point3D));
    
    
    // Create Streams for overlapping memcopy and compute
    int nStreams=deviceCount*2;
    hipStream_t* stream=(hipStream_t*)malloc(nStreams*sizeof(hipStream_t));;
    
    
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        for (int i = 0; i < 2; ++i){
            hipStreamCreate(&stream[i+dev*2]);
            
        }
    }
    cudaCheckErrors("Stream creation fail");

    int nangles_device=(nangles+deviceCount-1)/deviceCount;
    int nangles_last_device=(nangles-(deviceCount-1)*nangles_device);
    unsigned int noOfKernelCalls = (nangles_device+PROJ_PER_BLOCK-1)/PROJ_PER_BLOCK;  // We'll take care of bounds checking inside the loop if nalpha is not divisible by PROJ_PER_BLOCK
    unsigned int last_device_blocks= (nangles_last_device+PROJ_PER_BLOCK-1)/PROJ_PER_BLOCK; // we will use this in the memory management.
    int projection_this_block;
    hipTextureObject_t *texImg = new hipTextureObject_t[deviceCount];
    hipArray **d_cuArrTex = new hipArray*[deviceCount];
    
    for (unsigned int sp=0;sp<splits;sp++){
        
        // Create texture objects for all GPUs
        
        
        size_t linear_idx_start;
        //First one shoudl always be  the same size as all the rest but the last
        linear_idx_start= (size_t)sp*(size_t)geoArray[0].nVoxelX*(size_t)geoArray[0].nVoxelY*(size_t)geoArray[0].nVoxelZ;
        
        
        CreateTexture(deviceCount,&img[linear_idx_start],geoArray[sp],d_cuArrTex,texImg,!sp);
        cudaCheckErrors("Texture object creation fail");
        
        
        // Prepare kernel lauch variables
        
        int divU,divV;
        divU=PIXEL_SIZE_BLOCK;
        divV=PIXEL_SIZE_BLOCK;
        dim3 grid((geoArray[sp].nDetecU+divU-1)/divU,(geoArray[0].nDetecV+divV-1)/divV,1);
        dim3 block(divU,divV,PROJ_PER_BLOCK);
        
        unsigned int proj_global;
        unsigned int i;
        // Now that we have prepared the image (piece of image) and parameters for kernels
        // we project for all angles.
        for ( i=0; i<noOfKernelCalls; i++){
            for (dev=0;dev<deviceCount;dev++){
                hipSetDevice(dev);
                
                for(unsigned int j=0; j<PROJ_PER_BLOCK; j++){
                    proj_global=(i*PROJ_PER_BLOCK+j)+dev*nangles_device;
                    if (proj_global>=nangles)
                        break;
                    if ((i*PROJ_PER_BLOCK+j)>=nangles_device)
                        break;
                    geoArray[sp].alpha=angles[proj_global*3];
                    geoArray[sp].theta=angles[proj_global*3+1];
                    geoArray[sp].psi  =angles[proj_global*3+2];
                    
                    
                    //precomute distances for faster execution
                    //Precompute per angle constant stuff for speed
                    computeDeltas_Siddon(geoArray[sp],proj_global, &uvOrigin, &deltaU, &deltaV, &source);
                    //Ray tracing!
                    projParamsArrayHost[4*j]=uvOrigin;		// 6*j because we have 6 Point3D values per projection
                    projParamsArrayHost[4*j+1]=deltaU;
                    projParamsArrayHost[4*j+2]=deltaV;
                    projParamsArrayHost[4*j+3]=source;
                    
                }
                hipMemcpyToSymbolAsync(HIP_SYMBOL(projParamsArrayDev), projParamsArrayHost, sizeof(Point3D)*4*PROJ_PER_BLOCK,0,hipMemcpyHostToDevice,stream[dev*2]);
                hipStreamSynchronize(stream[dev*2]);
                cudaCheckErrors("kernel fail");
                kernelPixelDetector<<<grid,block,0,stream[dev*2]>>>(geoArray[sp],dProjection[(i%2)+dev*2],i,nangles_device,texImg[dev]);
            }
            // Now that the computation is happening, we need to either prepare the memory for
            // combining of the projections (splits>1) or start removing previous results.
            
            // How many projections are we dealing with in the current block??
            
            
            // If our image does not fit in memory then we need to make sure we accumulate previous results too.
            if( !fits_in_memory && sp>0 ){
                // First, grab previous results and put them in the auxiliary variable
                for (dev = 0; dev < deviceCount; dev++){
                    projection_this_block=PROJ_PER_BLOCK;
                    hipSetDevice(dev);
                    // this werid code makes sure we dont access bad memory. Its necesary for deviceCount>2
                    if (dev+1==deviceCount){ // if its the last device
                        if(i+1==last_device_blocks) // If we are in the last block of the last device, how many projections?
                            projection_this_block=nangles_last_device-(last_device_blocks-1)*PROJ_PER_BLOCK;
                        if(i+1>last_device_blocks) // As the last device can have less blocs, i may be over it.
                            break;
                    }else{
                        if(i+1==noOfKernelCalls) // if its not the last device, it can still be the lat block
                            projection_this_block=nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK;
                    }
                    hipMemcpyAsync(dProjection_accum[(i%2)+dev*2], result[i*PROJ_PER_BLOCK+dev*nangles_device], projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyHostToDevice,stream[dev*2+1]);
                }
                // Second, take the results from current compute call and add it to the code in execution.
                for (dev = 0; dev < deviceCount; dev++){
                    
                    projection_this_block=PROJ_PER_BLOCK;
                    hipSetDevice(dev);
                    // this werid code makes sure we dont access bad memory. Its necesary for deviceCount>2
                    if (dev+1==deviceCount){ // if its the last device
                        if(i+1==last_device_blocks) // If we are in the last block of the last device, how many projections?
                            projection_this_block=nangles_last_device-(last_device_blocks-1)*PROJ_PER_BLOCK;
                        if(i+1>last_device_blocks) // As the last device can have less blocs, i may be over it.
                            break;
                    }else{
                        if(i+1==noOfKernelCalls) // if its not the last device, it can still be the lat block
                            projection_this_block=nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK;
                    }
                    hipStreamSynchronize(stream[dev*2+1]); // wait until copy is finished
                    vecAddInPlace<<<(geo.nDetecU*geo.nDetecV*projection_this_block+MAXTREADS-1)/MAXTREADS,MAXTREADS,0,stream[dev*2]>>>(dProjection[(i%2)+dev*2],dProjection_accum[(i%2)+dev*2],(unsigned long)geo.nDetecU*geo.nDetecV*projection_this_block);
                }
            }
            
            // Now, lets get out the projections from the previous execution of the kernels.
            if (i>0){
                for (dev = 0; dev < deviceCount; dev++){
                    projection_this_block=PROJ_PER_BLOCK;
                    hipSetDevice(dev);
                    if (dev+1==deviceCount && i+1==noOfKernelCalls && last_device_blocks!=noOfKernelCalls){ 
                            projection_this_block=nangles_last_device-(last_device_blocks-1)*PROJ_PER_BLOCK;
                    }
                    hipMemcpyAsync(result[(i-1)*PROJ_PER_BLOCK+dev*nangles_device], dProjection[(int)(!(i%2))+dev*2],  projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyDeviceToHost,stream[dev*2+1]);
                }
            }
            // Make sure Computation on kernels has finished before we launch the next batch.
            for (dev = 0; dev < deviceCount; dev++){
                hipSetDevice(dev);
                hipStreamSynchronize(stream[dev*2]);
            }   
        }// END angles loop (i)
        
        
        // We still have the last one to get out, do that one
        
        int projection_this_block;
        for (dev = 0; dev < deviceCount; dev++){
            projection_this_block=PROJ_PER_BLOCK;
            hipSetDevice(dev);
            // this werid code makes sure we dont access bad memory. Its necesary for deviceCount>2
            if (dev+1==deviceCount){ // if its the last device
                projection_this_block=nangles_last_device-(last_device_blocks-1)*PROJ_PER_BLOCK;
                if(i>last_device_blocks) // As the last device can have less blocs, i may be over it.
                    break;
            }else{
               projection_this_block=nangles_device-(noOfKernelCalls-1)*PROJ_PER_BLOCK;
            }
            hipDeviceSynchronize();
            cudaCheckErrors("Fail memcopy fail");
            hipMemcpyAsync(result[(i-1)*PROJ_PER_BLOCK+dev*nangles_device], dProjection[(int)(!(i%2))+dev*2], projection_this_block*geo.nDetecV*geo.nDetecU*sizeof(float), hipMemcpyDeviceToHost,stream[dev*2+1]);
        }
        // Free memory for the next piece of image
       
        hipDeviceSynchronize();
    }//END image splits loop
     
    
    cudaCheckErrors("Main loop  fail");
    ///////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////
    for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipDestroyTextureObject(texImg[dev]);
            hipFreeArray(d_cuArrTex[dev]);
    }
    // Freeing Stage
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipFree(dProjection[dev*2]);
        hipFree(dProjection[dev*2+1]);
        
    }
    free(dProjection);
    
    if(!fits_in_memory){
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipFree(dProjection_accum[dev*2]);
            hipFree(dProjection_accum[dev*2+1]);
            
        }
        free(dProjection_accum);
    }
    freeGeoArray(splits,geoArray);
    hipHostFree(projParamsArrayHost);
   
    
    for (int i = 0; i < nStreams; ++i)
        hipStreamDestroy(stream[i]) ;
    
    if (isHostRegisterSupported & (splits>1 |deviceCount>1)){
        hipHostUnregister(img);
    }
    cudaCheckErrors("hipFree  fail");
    
    hipDeviceReset();
    return 0;
}




void CreateTexture(int num_devices,const float* imagedata,Geometry geo,hipArray** d_cuArrTex, hipTextureObject_t *texImage,bool alloc)
{
    //size_t size_image=geo.nVoxelX*geo.nVoxelY*geo.nVoxelZ;
    const hipExtent extent = make_hipExtent(geo.nVoxelX, geo.nVoxelY, geo.nVoxelZ);
    if(alloc){
    for (unsigned int i = 0; i < num_devices; i++){
        hipSetDevice(i);
        
        //hipArray Descriptor
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        //cuda Array
        hipMalloc3DArray(&d_cuArrTex[i], &channelDesc, extent);
    }
    }
    for (unsigned int i = 0; i < num_devices; i++){
        hipSetDevice(i);
        hipMemcpy3DParms copyParams = {0};
        //Array creation
        copyParams.srcPtr   = make_hipPitchedPtr((void *)imagedata, extent.width*sizeof(float), extent.width, extent.height);
        copyParams.dstArray = d_cuArrTex[i];
        copyParams.extent   = extent;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3DAsync(&copyParams);
    }
    for (unsigned int i = 0; i < num_devices; i++){
        hipSetDevice(i);
        hipResourceDesc    texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array  = d_cuArrTex[i];
        hipTextureDesc     texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        texDescr.filterMode = hipFilterModePoint;
        texDescr.addressMode[0] = hipAddressModeBorder;
        texDescr.addressMode[1] = hipAddressModeBorder;
        texDescr.addressMode[2] = hipAddressModeBorder;
        texDescr.readMode = hipReadModeElementType;
        hipCreateTextureObject(&texImage[i], &texRes, &texDescr, NULL);
        
    }
    for (unsigned int i = 0; i < num_devices; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
    }
    cudaCheckErrors("Texture object creation fail");
}

/* This code generates the geometries needed to split the image properly in
 * cases where the entire image does not fit in the memory of the GPU
 **/
void splitImage(unsigned int splits,Geometry geo,Geometry* geoArray, unsigned int nangles){
    
    unsigned long splitsize=(geo.nVoxelZ+splits-1)/splits;// ceil if not divisible
    for(unsigned int sp=0;sp<splits;sp++){
        geoArray[sp]=geo;
        // All of them are splitsize, but the last one, possible
        geoArray[sp].nVoxelZ=((sp+1)*splitsize<geo.nVoxelZ)?  splitsize:  geo.nVoxelZ-splitsize*sp;
        geoArray[sp].sVoxelZ= geoArray[sp].nVoxelZ* geoArray[sp].dVoxelZ;
        
        // We need to redefine the offsets, as now each subimage is not aligned in the origin.
        geoArray[sp].offOrigZ=(float *)malloc(nangles*sizeof(float));
        for (unsigned int i=0;i<nangles;i++){
            geoArray[sp].offOrigZ[i]=geo.offOrigZ[i]-geo.sVoxelZ/2+sp*geoArray[0].sVoxelZ+geoArray[sp].sVoxelZ/2;
        }
        
    }
    
}

/* This code precomputes The location of the source and the Delta U and delta V (in the warped space)
 * to compute the locations of the x-rays. While it seems verbose and overly-optimized,
 * it does saves about 30% of each of the kernel calls. Thats something!
 **/
void computeDeltas_Siddon(Geometry geo,int i, Point3D* uvorigin, Point3D* deltaU, Point3D* deltaV, Point3D* source){
    Point3D S;
    S.x=geo.DSO[i];
    S.y=0;
    S.z=0;
    
    //End point
    Point3D P,Pu0,Pv0;
    
    P.x  =-(geo.DSD[i]-geo.DSO[i]);   P.y  = geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       P.z  = geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pu0.x=-(geo.DSD[i]-geo.DSO[i]);   Pu0.y= geo.dDetecU*(1-((float)geo.nDetecU/2)+0.5);       Pu0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pv0.x=-(geo.DSD[i]-geo.DSO[i]);   Pv0.y= geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       Pv0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-1);
    // Geomtric trasnformations:
    // Now we have the Real world (OXYZ) coordinates of the bottom corner and its two neighbours.
    // The obkjective is to get a position of the detector in a coordinate system where:
    // 1-units are voxel size (in each direction can be different)
    // 2-The image has the its first voxel at (0,0,0)
    // 3-The image never rotates
    
    // To do that, we need to compute the "deltas" the detector, or "by how much
    // (in new xyz) does the voxels change when and index is added". To do that
    // several geometric steps needs to be changed
    
    //1.Roll,pitch,jaw
    // The detector can have a small rotation.
    // according to
    //"A geometric calibration method for cone beam CT systems" Yang K1, Kwan AL, Miller DF, Boone JM. Med Phys. 2006 Jun;33(6):1695-706.
    // Only the Z rotation will have a big influence in the image quality when they are small.
    // Still all rotations are supported
    
    // To roll pitch jaw, the detector has to be in centered in OXYZ.
    P.x=0;Pu0.x=0;Pv0.x=0;
    
    // Roll pitch yaw
    rollPitchYaw(geo,i,&P);
    rollPitchYaw(geo,i,&Pu0);
    rollPitchYaw(geo,i,&Pv0);
    //Now ltes translate the points where they shoudl be:
    P.x=P.x-(geo.DSD[i]-geo.DSO[i]);
    Pu0.x=Pu0.x-(geo.DSD[i]-geo.DSO[i]);
    Pv0.x=Pv0.x-(geo.DSD[i]-geo.DSO[i]);
    
    //1: Offset detector
    
    
    //S doesnt need to chagne
    
    
    //3: Rotate (around z)!
    Point3D Pfinal, Pfinalu0, Pfinalv0;
    Pfinal.x  =P.x;
    Pfinal.y  =P.y  +geo.offDetecU[i]; Pfinal.z  =P.z  +geo.offDetecV[i];
    Pfinalu0.x=Pu0.x;
    Pfinalu0.y=Pu0.y  +geo.offDetecU[i]; Pfinalu0.z  =Pu0.z  +geo.offDetecV[i];
    Pfinalv0.x=Pv0.x;
    Pfinalv0.y=Pv0.y  +geo.offDetecU[i]; Pfinalv0.z  =Pv0.z  +geo.offDetecV[i];
    
    eulerZYZ(geo,&Pfinal);
    eulerZYZ(geo,&Pfinalu0);
    eulerZYZ(geo,&Pfinalv0);
    eulerZYZ(geo,&S);
    
    //2: Offset image (instead of offseting image, -offset everything else)
    
    Pfinal.x  =Pfinal.x-geo.offOrigX[i];     Pfinal.y  =Pfinal.y-geo.offOrigY[i];     Pfinal.z  =Pfinal.z-geo.offOrigZ[i];
    Pfinalu0.x=Pfinalu0.x-geo.offOrigX[i];   Pfinalu0.y=Pfinalu0.y-geo.offOrigY[i];   Pfinalu0.z=Pfinalu0.z-geo.offOrigZ[i];
    Pfinalv0.x=Pfinalv0.x-geo.offOrigX[i];   Pfinalv0.y=Pfinalv0.y-geo.offOrigY[i];   Pfinalv0.z=Pfinalv0.z-geo.offOrigZ[i];
    S.x=S.x-geo.offOrigX[i];               S.y=S.y-geo.offOrigY[i];               S.z=S.z-geo.offOrigZ[i];
    
    // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
    Pfinal.x  =Pfinal.x+geo.sVoxelX/2;      Pfinal.y  =Pfinal.y+geo.sVoxelY/2;          Pfinal.z  =Pfinal.z  +geo.sVoxelZ/2;
    Pfinalu0.x=Pfinalu0.x+geo.sVoxelX/2;    Pfinalu0.y=Pfinalu0.y+geo.sVoxelY/2;        Pfinalu0.z=Pfinalu0.z+geo.sVoxelZ/2;
    Pfinalv0.x=Pfinalv0.x+geo.sVoxelX/2;    Pfinalv0.y=Pfinalv0.y+geo.sVoxelY/2;        Pfinalv0.z=Pfinalv0.z+geo.sVoxelZ/2;
    S.x      =S.x+geo.sVoxelX/2;          S.y      =S.y+geo.sVoxelY/2;              S.z      =S.z      +geo.sVoxelZ/2;
    
    //4. Scale everything so dVoxel==1
    Pfinal.x  =Pfinal.x/geo.dVoxelX;      Pfinal.y  =Pfinal.y/geo.dVoxelY;        Pfinal.z  =Pfinal.z/geo.dVoxelZ;
    Pfinalu0.x=Pfinalu0.x/geo.dVoxelX;    Pfinalu0.y=Pfinalu0.y/geo.dVoxelY;      Pfinalu0.z=Pfinalu0.z/geo.dVoxelZ;
    Pfinalv0.x=Pfinalv0.x/geo.dVoxelX;    Pfinalv0.y=Pfinalv0.y/geo.dVoxelY;      Pfinalv0.z=Pfinalv0.z/geo.dVoxelZ;
    S.x      =S.x/geo.dVoxelX;          S.y      =S.y/geo.dVoxelY;            S.z      =S.z/geo.dVoxelZ;
    
    
    //mexPrintf("COR: %f \n",geo.COR[i]);
    //5. apply COR. Wherever everything was, now its offesetd by a bit
    float CORx, CORy;
    CORx=-geo.COR[i]*sin(geo.alpha)/geo.dVoxelX;
    CORy= geo.COR[i]*cos(geo.alpha)/geo.dVoxelY;
    Pfinal.x+=CORx;   Pfinal.y+=CORy;
    Pfinalu0.x+=CORx;   Pfinalu0.y+=CORy;
    Pfinalv0.x+=CORx;   Pfinalv0.y+=CORy;
    S.x+=CORx; S.y+=CORy;
    
    // return
    
    *uvorigin=Pfinal;
    
    deltaU->x=Pfinalu0.x-Pfinal.x;
    deltaU->y=Pfinalu0.y-Pfinal.y;
    deltaU->z=Pfinalu0.z-Pfinal.z;
    
    deltaV->x=Pfinalv0.x-Pfinal.x;
    deltaV->y=Pfinalv0.y-Pfinal.y;
    deltaV->z=Pfinalv0.z-Pfinal.z;
    
    *source=S;
}


#ifndef PROJECTION_HPP

float maxDistanceCubeXY(Geometry geo, float alpha,int i){
    ///////////
    // Compute initial "t" so we access safely as less as out of bounds as possible.
    //////////
    
    
    float maxCubX,maxCubY;
    // Forgetting Z, compute max distance: diagonal+offset
    maxCubX=(geo.sVoxelX/2+ abs(geo.offOrigX[i]))/geo.dVoxelX;
    maxCubY=(geo.sVoxelY/2+ abs(geo.offOrigY[i]))/geo.dVoxelY;
    
    return geo.DSO[i]/geo.dVoxelX-sqrt(maxCubX*maxCubX+maxCubY*maxCubY);
    
}
void rollPitchYaw(Geometry geo,int i, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=cos(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) - sin(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) + sin(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->y=sin(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) + cos(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) - cos(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->z=-sin(geo.dPitch[i])*auxPoint.x
            +cos(geo.dPitch[1])*sin(geo.dYaw[i])*auxPoint.y
            +cos(geo.dPitch[1])*cos(geo.dYaw[i])*auxPoint.z;
    
}
void eulerZYZ(Geometry geo, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=(+cos(geo.alpha)*cos(geo.theta)*cos(geo.psi)-sin(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-cos(geo.alpha)*cos(geo.theta)*sin(geo.psi)-sin(geo.alpha)*cos(geo.psi))*auxPoint.y+
            cos(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->y=(+sin(geo.alpha)*cos(geo.theta)*cos(geo.psi)+cos(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-sin(geo.alpha)*cos(geo.theta)*sin(geo.psi)+cos(geo.alpha)*cos(geo.psi))*auxPoint.y+
            sin(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->z=-sin(geo.theta)*cos(geo.psi)*auxPoint.x+
            sin(geo.theta)*sin(geo.psi)*auxPoint.y+
            cos(geo.theta)*auxPoint.z;
    
    
}
//______________________________________________________________________________
//
//      Function:       freeGeoArray
//
//      Description:    Frees the memory from the geometry array for multiGPU.
//______________________________________________________________________________
void freeGeoArray(unsigned int splits,Geometry* geoArray){
    for(unsigned int sp=0;sp<splits;sp++){
        free(geoArray[sp].offOrigZ);
    }
    free(geoArray);
}
//______________________________________________________________________________
//
//      Function:       checkFreeMemory
//
//      Description:    check available memory on devices
//______________________________________________________________________________
void checkFreeMemory(int deviceCount,size_t *mem_GPU_global){
    size_t memfree;
    size_t memtotal;
    
    for (int dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipMemGetInfo(&memfree,&memtotal);
        if(dev==0) *mem_GPU_global=memfree;
        if(memfree<memtotal/2){
            mexErrMsgIdAndTxt("Ax:Siddon_projection:GPUmemory","One (or more) of your GPUs is being heavily used by another program (possibly graphics-based).\n Free the GPU to run TIGRE\n");
        }
        cudaCheckErrors("Check mem error");
        
        *mem_GPU_global=(memfree<*mem_GPU_global)?memfree:*mem_GPU_global;
    }
    *mem_GPU_global=(size_t)((double)*mem_GPU_global*0.95);
    
    //*mem_GPU_global= insert your known number here, in bytes.
}
#endif
