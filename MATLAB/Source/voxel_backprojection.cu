#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA function for backrpojection using FDK weigts for CBCT
 *
 *
 * CODE by  Ander Biguri
 *          Optimized and modified by RB
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */

#define  PI_2 1.57079632679489661923
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "voxel_backprojection.hpp"
#include "mex.h"
#include <math.h>

// https://stackoverflow.com/questions/16282136/is-there-a-cuda-equivalent-of-perror
#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("CBCT:CUDA:Atb",hipGetErrorString(__err));\
        } \
} while (0)
    
    
#define MAXTREADS 1024
    /*GEOMETRY DEFINITION
     *
     *                Detector plane, behind
     *            |-----------------------------|
     *            |                             |
     *            |                             |
     *            |                             |
     *            |                             |
     *            |      +--------+             |
     *            |     /        /|             |
     *   A Z      |    /        / |*D           |
     *   |        |   +--------+  |             |
     *   |        |   |        |  |             |
     *   |        |   |     *O |  +             |
     *   *--->y   |   |        | /              |
     *  /         |   |        |/               |
     * V X        |   +--------+                |
     *            |-----------------------------|
     *
     *           *S
     *
     *
     *
     *
     *
     **/
    
    void CreateTexture(int num_devices,float* projectiondata,Geometry geo,hipArray** d_cuArrTex,unsigned int nangles, hipTextureObject_t *texImage,hipStream_t* stream, int nStreamDevice,bool allocate);


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// RB, 10/31/2016: Add constant memory arrays to store parameters for all projections to be analyzed during a single kernel call
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// The optimal values of two constants obtained by RB on NVIDIA Quadro K2200 (4 GB RAM, 640 CUDA cores) for 512^3 volume and 512^3 projections (512 proj, each 512 x 512) were:
// PROJ_PER_KERNEL = 32 or 16 (very similar times)
// VOXELS_PER_THREAD = 8
// Speedup of the entire FDK backprojection (not only kernel run, also memcpy etc.) was nearly 4x relative to the original (single projection, single voxel per thread) code.
// (e.g. 16.2 s vs. ~62 s).

const int PROJ_PER_KERNEL = 32;  // Number of 2D projections to be analyzed by a single thread. This can be tweaked to see what works best. 32 was the optimal value in the paper by Zinsser and Keck.
const int VOXELS_PER_THREAD = 8;  // Number of voxels to be computed by s single thread. Can be tweaked to see what works best. 4 was the optimal value in the paper by Zinsser and Keck.

// We have PROJ_PER_KERNEL projections and we need 6 parameters for each projection:
//   deltaX, deltaY, deltaZ, xyzOrigin, offOrig, offDetec
// So we need to keep PROJ_PER_KERNEL*6 values in our deltas array FOR EACH CALL to our main kernel
// (they will be updated in the main loop before each kernel call).

__constant__ Point3D projParamsArrayDev[6*PROJ_PER_KERNEL];  // Dev means it is on device

// We also need a corresponding array on the host side to be filled before each kernel call, then copied to the device (array in constant memory above)
// Point3D projParamsArrayHost[6*PROJ_PER_KERNEL];   // Host means it is host memory

// Now we also need to store sinAlpha and cosAlpha for each projection (two floats per projection)
__constant__ float projSinCosArrayDev[5*PROJ_PER_KERNEL];



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// END RB, 10/31/2016: Add constant memory arrays to store parameters for all projections to be analyzed during a single kernel call
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


//______________________________________________________________________________
//
//      Function:       kernelPixelBackprojectionFDK
//
//      Description:    Main FDK backprojection kernel
//______________________________________________________________________________

__global__ void kernelPixelBackprojectionFDK(const Geometry geo, float* image,const int currProjSetNumber, const int totalNoOfProjections, hipTextureObject_t tex)
{
    
    // Old kernel call signature:
    // kernelPixelBackprojectionFDK<<<grid,block>>>(geo,dimage,i,deltaX,deltaY,deltaZ,xyzOrigin,offOrig,offDetec,sinalpha,cosalpha);
    // We just read in most of the params from the constant memory instead of getting them from the param list.
    // This is because we now have MANY params, since single kernel processes more than one projection!
    /* __global__ void kernelPixelBackprojectionFDK(const Geometry geo,
     * float* image,
     * const int indAlpha,
     * const Point3D deltaX ,
     * const Point3D deltaY,
     * const Point3D deltaZ,
     * const Point3D xyzOrigin,
     * const Point3D xyzOffset,
     * const Point3D uv0Offset,
     * const float sinalpha,
     * const float cosalpha){
     */
    unsigned long indY = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long indX = blockIdx.x * blockDim.x + threadIdx.x;
    // unsigned long startIndZ = blockIdx.z * blockDim.z + threadIdx.z;  // This is only STARTING z index of the column of voxels that the thread will handle
    unsigned long startIndZ = blockIdx.z * VOXELS_PER_THREAD + threadIdx.z;  // This is only STARTING z index of the column of voxels that the thread will handle
    //Make sure we don't go out of bounds
    if (indX>=geo.nVoxelX | indY>=geo.nVoxelY |startIndZ>=geo.nVoxelZ)
        return;
    
    // We'll keep a local auxiliary array of values of a column of voxels that this thread will update
    float voxelColumn[VOXELS_PER_THREAD];
    
    // First we need to copy the curent 3D volume values from the column to our auxiliary array so that we can then
    // work on them (update them by computing values from multiple projections) locally - avoiding main memory reads/writes
    
    int colIdx;
#pragma unroll
    for(colIdx=0; colIdx<VOXELS_PER_THREAD; colIdx++)
    {
        unsigned long indZ = startIndZ + colIdx;
        // If we are out of bounds, break the loop. The voxelColumn array will be updated partially, but it is OK, because we won't
        // be trying to copy the out of bounds values back to the 3D volume anyway (bounds checks will be done in the final loop where the updated values go back to the main volume)
        if(indZ>=geo.nVoxelZ)
            break;   // break the loop.
        
        unsigned long long idx =indZ*geo.nVoxelX*geo.nVoxelY+indY*geo.nVoxelX + indX;
        voxelColumn[colIdx] = image[idx];   // Read the current volume value that we'll update by computing values from MULTIPLE projections (not just one)
        // We'll be updating the local (register) variable, avoiding reads/writes from the slow main memory.
    }  // END copy 3D volume voxels to local array
    
    // Now iterate through projections
#pragma unroll
    for(int projNumber=0; projNumber<PROJ_PER_KERNEL; projNumber++)
    {
        // Get the current parameters from parameter arrays in constant memory.
        int indAlpha = currProjSetNumber*PROJ_PER_KERNEL+projNumber;  // This is the ABSOLUTE projection number in the projection array
        
        // Our currImageVal will be updated by hovewer many projections we had left in the "remainder" - that's OK.
        if(indAlpha>=totalNoOfProjections)
            break;
        
        Point3D deltaX = projParamsArrayDev[6*projNumber];  // 6*projNumber because we have 6 Point3D values per projection
        Point3D deltaY = projParamsArrayDev[6*projNumber+1];
        Point3D deltaZ = projParamsArrayDev[6*projNumber+2];
        Point3D xyzOrigin = projParamsArrayDev[6*projNumber+3];
        Point3D xyzOffset = projParamsArrayDev[6*projNumber+4];
        Point3D S = projParamsArrayDev[6*projNumber+5];
        
        float sinalpha = projSinCosArrayDev[5*projNumber];     // 2*projNumber because we have 2 float (sin or cos angle) values per projection
        float cosalpha = projSinCosArrayDev[5*projNumber+1];
        float COR = projSinCosArrayDev[5*projNumber+2];
        float DSD = projSinCosArrayDev[5*projNumber+3];
        float DSO = projSinCosArrayDev[5*projNumber+4];
        
        float auxCOR=COR/geo.dDetecU;
        // Now iterate through Z in our voxel column FOR A GIVEN PROJECTION
#pragma unroll
        for(colIdx=0; colIdx<VOXELS_PER_THREAD; colIdx++)
        {
            unsigned long indZ = startIndZ + colIdx;
            
            // If we are out of bounds, break the loop. The voxelColumn array will be updated partially, but it is OK, because we won't
            // be trying to copy the out of bounds values anyway (bounds checks will be done in the final loop where the values go to the main volume)
            if(indZ>=geo.nVoxelZ)
                break;   // break the loop.
            
            // "XYZ" in the scaled coordinate system of the current point. The image is rotated with the projection angles.
            Point3D P;
            P.x=(xyzOrigin.x+indX*deltaX.x+indY*deltaY.x+indZ*deltaZ.x);
            P.y=(xyzOrigin.y+indX*deltaX.y+indY*deltaY.y+indZ*deltaZ.y)-auxCOR;
            P.z=(xyzOrigin.z+indX*deltaX.z+indY*deltaY.z+indZ*deltaZ.z);
            
            // This is the vector defining the line from the source to the Voxel
            float vectX,vectY,vectZ;
            vectX=(P.x -S.x);
            vectY=(P.y -S.y);
            vectZ=(P.z -S.z);
            
            // Get the coordinates in the detector UV where the mid point of the voxel is projected.
            float t=__fdividef(DSO-DSD-S.x,vectX);
            float y,z;
            y=vectY*t+S.y;
            z=vectZ*t+S.z;
            float u,v;
            u=y+(float)geo.nDetecU*0.5f;
            v=z+(float)geo.nDetecV*0.5f;
            
            float weigth;
            float realx,realy;
            realx=-(geo.sVoxelX+geo.dVoxelX)*0.5f  +indX*geo.dVoxelX   +xyzOffset.x;
            realy=-(geo.sVoxelY+geo.dVoxelY)*0.5f  +indY*geo.dVoxelY   +xyzOffset.y+COR;
            
            weigth=__fdividef(DSO+realy*sinalpha-realx*cosalpha,DSO);
            
            weigth=__frcp_rd(weigth*weigth);
            
            // Get Value in the computed (U,V) and multiply by the corresponding weigth.
            // indAlpha is the ABSOLUTE number of projection in the projection array (NOT the current number of projection set!)
            
            voxelColumn[colIdx]+=tex3D<float>(tex, v, u ,indAlpha+0.5f)*weigth;
        }  // END iterating through column of voxels
        
    }  // END iterating through multiple projections
    
    // And finally copy the updated local voxelColumn array back to our 3D volume (main memory)
#pragma unroll
    for(colIdx=0; colIdx<VOXELS_PER_THREAD; colIdx++)
    {
        unsigned long indZ = startIndZ + colIdx;
        // If we are out of bounds, break the loop. The voxelColumn array will be updated partially, but it is OK, because we won't
        // be trying to copy the out of bounds values back to the 3D volume anyway (bounds checks will be done in the final loop where the values go to the main volume)
        if(indZ>=geo.nVoxelZ)
            break;   // break the loop.
        
        unsigned long long idx =indZ*geo.nVoxelX*geo.nVoxelY+indY*geo.nVoxelX + indX;
        image[idx] = voxelColumn[colIdx];   // Read the current volume value that we'll update by computing values from MULTIPLE projections (not just one)
        // We'll be updating the local (register) variable, avoiding reads/writes from the slow main memory.
        // According to references (Papenhausen), doing = is better than +=, since += requires main memory read followed by a write.
        // We did all the reads into the local array at the BEGINNING of this kernel. According to Papenhausen, this type of read-write split is
        // better for avoiding memory congestion.
    }  // END copy updated voxels from local array to our 3D volume
    
}  // END kernelPixelBackprojectionFDK




//______________________________________________________________________________
//
//      Function:       voxel_backprojection
//
//      Description:    Main host function for FDK backprojection (invokes the kernel)
//______________________________________________________________________________

int voxel_backprojection(float  *  projections, Geometry geo, float* result,float const * const alphas, int nalpha)
{
    
    
    
    // Prepare for MultiGPU
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    cudaCheckErrors("Device query fail");
    if (deviceCount == 0) {
        mexErrMsgIdAndTxt("Atb:Voxel_backprojection:GPUselect","There are no available device(s) that support CUDA\n");
    }
    
    // Check the available devices, and if they are the same
    int dev;
    checkDevices();
    
    // Split the CT problem
    unsigned int split_image;
    unsigned int split_projections;
    splitCTbackprojection(deviceCount,geo,nalpha,&split_image,&split_projections);
    
    
    cudaCheckErrors("Error");
    //Pagelock memory for synchronous copy.
    // Lets try to make the host memory pinned:
    // We laredy queried the GPU and assuemd they are the same, thus should have the same attributes.
    int isHostRegisterSupported;
    hipDeviceGetAttribute(&isHostRegisterSupported,hipDeviceAttributeHostRegisterSupported,0);
    // empirical testing shows that when the image split is smaller than 1 (also implies the image is not very big), the time to
    // pin the memory is greater than the lost time in Synchronously launching the memcpys. This is only worth it when the image is too big.
    if (isHostRegisterSupported & (split_image>1 |deviceCount>1)){
        hipHostRegister(result, (size_t)geo.nVoxelX*(size_t)geo.nVoxelY*(size_t)geo.nVoxelZ*(size_t)sizeof(float),hipHostRegisterPortable);
    }
//     if (isHostRegisterSupported ){
//         hipHostRegister(projections, (size_t)geo.nDetecU*(size_t)geo.nDetecV*(size_t)nalpha*(size_t)sizeof(float),hipHostRegisterPortable);
//     }
    cudaCheckErrors("Error pinning memory");
    
    
    // Create the arrays for the geometry. The main difference is that geo.offZ has been tuned for the
    // image slices. The rest of the Geometry is the same
    Geometry* geoArray=(Geometry*)malloc(split_image*deviceCount*sizeof(Geometry));
    createGeoArray(split_image*deviceCount,geo,geoArray,nalpha);
    
    // Now lest allocate all the image memory on the GPU, so we can use it later. If we have made our numbers correctly
    // in the previous section this should leave enough space for the textures.
    size_t num_bytes_img = (size_t)geo.nVoxelX*(size_t)geo.nVoxelY*(size_t)geoArray[0].nVoxelZ* sizeof(float);
    float** dimage=(float**)malloc(deviceCount*sizeof(float*));
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipMalloc((void**)&dimage[dev], num_bytes_img);
        cudaCheckErrors("hipMalloc fail");
    }
    
    //If it is the first time, lets make sure our image is zeroed.
    int nStreamDevice=2;
    int nStreams=deviceCount*nStreamDevice;
    hipStream_t* stream=(hipStream_t*)malloc(nStreams*sizeof(hipStream_t));;
    
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        for (int i = 0; i < nStreamDevice; ++i){
            hipStreamCreate(&stream[i+dev*nStreamDevice]);
            
        }
    }
    

     
    
    // Kernel auxiliary variables
    Point3D* projParamsArrayHost;
    hipHostMalloc((void**)&projParamsArrayHost,6*PROJ_PER_KERNEL*sizeof(Point3D));
    float* projSinCosArrayHost;
    hipHostMalloc((void**)&projSinCosArrayHost,5*PROJ_PER_KERNEL*sizeof(float));
    
    
    // Texture object variables
    hipTextureObject_t *texProj;
    hipArray **d_cuArrTex;
    texProj =(hipTextureObject_t*)malloc(deviceCount*2*sizeof(hipTextureObject_t));
    d_cuArrTex =(hipArray**)malloc(deviceCount*2*sizeof(hipArray*));
    
    // Auxiliary Host page-locked memory for fast and asycnornous memcpy.

    // Start with the main loop. The Projection data needs to be allocated and dealocated in the main loop
    // as due to the nature of cudaArrays, we can not reuse them. This should not be a problem for the fast execution
    // of the code, as repeated allocation and deallocation only happens when the projection data is very very big,
    // and therefore allcoation time should be negligible, fluctuation of other computations should mask the time.
    unsigned long long proj_linear_idx_start;
    unsigned int proj_split_overlap_number;
    unsigned int current_proj_split_size,current_proj_overlap_split_size;
    size_t num_bytes_img_curr;
    size_t img_linear_idx_start;
    float** partial_projection;
    size_t* proj_split_size;
    
    
    
    for(unsigned int img_slice=0;img_slice<split_image;img_slice++){
        // Initialize the memory if its the first time.
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipMemset(dimage[dev],0,num_bytes_img);
            cudaCheckErrors("memset fail");
        }
        
        for( unsigned int proj=0;proj<split_projections;proj++){
            
            
            // What is the size of the current chunk of proejctions we need in?
            current_proj_split_size=(nalpha+split_projections-1)/split_projections;
            // if its the last one its probably less
            current_proj_split_size=((proj+1)*current_proj_split_size<nalpha)?  current_proj_split_size:  nalpha-current_proj_split_size*proj;
            
            // We are going to split it in the same amount of kernels we need to execute.
            proj_split_overlap_number=(current_proj_split_size+PROJ_PER_KERNEL-1)/PROJ_PER_KERNEL;
            
            // Create pointer to pointers of projections and precompute their location and size.
            if(!proj && !img_slice){
                partial_projection=(float**)malloc(proj_split_overlap_number*sizeof(float*));
                proj_split_size=(size_t*)malloc(proj_split_overlap_number*sizeof(size_t*));
            }
            for(unsigned int proj_block_split=0; proj_block_split<proj_split_overlap_number;proj_block_split++){
                // Crop the last one, as its likely its not completely divisible.
                // now lets split this for simultanoeus memcopy and compute.
                // We want to make sure that if we can, we run PROJ_PER_KERNEL projections, to maximize kernel acceleration
                // current_proj_overlap_split_size units = angles
                current_proj_overlap_split_size=max((current_proj_split_size+proj_split_overlap_number-1)/proj_split_overlap_number,PROJ_PER_KERNEL);
                current_proj_overlap_split_size=(proj_block_split<proj_split_overlap_number-1)?current_proj_overlap_split_size:current_proj_split_size-(proj_split_overlap_number-1)*current_proj_overlap_split_size;
                //Get the linear index where the current memory chunk starts.
                
                proj_linear_idx_start=(unsigned long long)((nalpha+split_projections-1)/split_projections)*(unsigned long long)proj*(unsigned long long)geo.nDetecU*(unsigned long long)geo.nDetecV;
                proj_linear_idx_start+=proj_block_split*max((current_proj_split_size+proj_split_overlap_number-1)/proj_split_overlap_number,PROJ_PER_KERNEL)*(unsigned long long)geo.nDetecU*(unsigned long long)geo.nDetecV;
                //Store result
                proj_split_size[proj_block_split]=current_proj_overlap_split_size;
                partial_projection[proj_block_split]=&projections[proj_linear_idx_start];
                
            }                
            for(unsigned int proj_block_split=0; proj_block_split<proj_split_overlap_number;proj_block_split++){

                // Now get the projections on memory

                CreateTexture(deviceCount,
                        partial_projection[proj_block_split],geo,
                        &d_cuArrTex[(proj_block_split%2)*deviceCount],
                        proj_split_size[proj_block_split],
                        &texProj   [(proj_block_split%2)*deviceCount],
                        stream, nStreamDevice,
                        (proj_block_split<2)&!proj&!img_slice);// Only allocate if its the first 2 calls
                
                for (dev = 0; dev < deviceCount; dev++){
                    hipSetDevice(dev);
                    hipStreamSynchronize(stream[dev*nStreamDevice+1]);
                 }
                               
                // Pin the next chunk of projection data, unpin the current one.
                for (dev = 0; dev < deviceCount; dev++){
                    //Safety:
                    // Depends on the amount of GPUs, the case where a image slice is zero hight can happen.
                    // Just break the loop if we reached that point
                    if(geoArray[img_slice*deviceCount+dev].nVoxelZ==0)
                        break;
                    
                    hipSetDevice(dev);
                    
                    
                    
                    int divx,divy,divz;
                    // RB: Use the optimal (in their tests) block size from paper by Zinsser and Keck (16 in x and 32 in y).
                    // I tried different sizes and shapes of blocks (tiles), but it does not appear to significantly affect trhoughput, so
                    // let's stick with the values from Zinsser and Keck.
                    divx=16;
                    divy=32;
                    divz=VOXELS_PER_THREAD;      // We now only have 32 x 16 threads per block (flat tile, see below), BUT each thread works on a Z column of VOXELS_PER_THREAD voxels, so we effectively need fewer blocks!
                    
                    
                    dim3 grid((geo.nVoxelX+divx-1)/divx,
                            (geo.nVoxelY+divy-1)/divy,
                            (geoArray[img_slice*deviceCount+dev].nVoxelZ+divz-1)/divz);
                    
                    dim3 block(divx,divy,1);    // Note that we have 1 in the Z size, not divz, since each thread works on a vertical set of VOXELS_PER_THREAD voxels (so we only need a "flat" tile of threads, with depth of 1)
                    //////////////////////////////////////////////////////////////////////////////////////
                    // Main reconstruction loop: go through projections (rotation angles) and backproject
                    //////////////////////////////////////////////////////////////////////////////////////
                    
                    // Since we'll have multiple projections processed by a SINGLE kernel call, compute how many
                    // kernel calls we'll need altogether.
                    unsigned int noOfKernelCalls = (proj_split_size[proj_block_split]+PROJ_PER_KERNEL-1)/PROJ_PER_KERNEL;  // We'll take care of bounds checking inside the loop if nalpha is not divisible by PROJ_PER_KERNEL
                    for (unsigned int i=0; i<noOfKernelCalls; i++){
                        
                        // Now we need to generate and copy all data for PROJ_PER_KERNEL projections to constant memory so that our kernel can use it
                        unsigned int j;
                        for(j=0; j<PROJ_PER_KERNEL; j++){
                            
                            unsigned int currProjNumber_slice=i*PROJ_PER_KERNEL+j;
                            unsigned int currProjNumber_global=i*PROJ_PER_KERNEL+j                                                                          // index within kernel
                                                               +proj*(nalpha+split_projections-1)/split_projections                                          // index of the global projection split
                                                               +proj_block_split*max(current_proj_split_size/proj_split_overlap_number,PROJ_PER_KERNEL); // indexof overlap current split
                            
                            if(currProjNumber_slice>=proj_split_size[proj_block_split])
                                break;  // Exit the loop. Even when we leave the param arrays only partially filled, this is OK, since the kernel will check bounds anyway.
                            if(currProjNumber_global>=nalpha)
                                break;  // Exit the loop. Even when we leave the param arrays only partially filled, this is OK, since the kernel will check bounds anyway.
                            
                            Point3D deltaX,deltaY,deltaZ,xyzOrigin, offOrig, /*offDetec,*/source;
                            float sinalpha,cosalpha;
                            
                            geoArray[img_slice*deviceCount+dev].alpha=-alphas[currProjNumber_global*3];//we got 3 angles now.
                            geoArray[img_slice*deviceCount+dev].theta=-alphas[currProjNumber_global*3+1];
                            geoArray[img_slice*deviceCount+dev].psi  =-alphas[currProjNumber_global*3+2];
                            
//                             mexPrintf("%u %f \n",i,geoArray[img_slice*deviceCount+dev].alpha);
//                             mexPrintf("%u \n",currProjNumber_global);
                            
                            sinalpha=sin(geoArray[img_slice*deviceCount+dev].alpha);
                            cosalpha=cos(geoArray[img_slice*deviceCount+dev].alpha);
                            
                            projSinCosArrayHost[5*j]=sinalpha;  // 2*j because we have 2 float (sin or cos angle) values per projection
                            projSinCosArrayHost[5*j+1]=cosalpha;
                            projSinCosArrayHost[5*j+2]=geo.COR[currProjNumber_global];
                            projSinCosArrayHost[5*j+3]=geo.DSD[currProjNumber_global];
                            projSinCosArrayHost[5*j+4]=geo.DSO[currProjNumber_global];
                            
                            computeDeltasCube(geoArray[img_slice*deviceCount+dev],currProjNumber_global,&xyzOrigin,&deltaX,&deltaY,&deltaZ,&source);
                            
                            offOrig.x=geo.offOrigX[currProjNumber_global];
                            offOrig.y=geo.offOrigY[currProjNumber_global];
                            offOrig.z=geoArray[img_slice*deviceCount+dev].offOrigZ[currProjNumber_global];
                            
                            projParamsArrayHost[6*j]=deltaX;		// 6*j because we have 6 Point3D values per projection
                            projParamsArrayHost[6*j+1]=deltaY;
                            projParamsArrayHost[6*j+2]=deltaZ;
                            projParamsArrayHost[6*j+3]=xyzOrigin;
                            projParamsArrayHost[6*j+4]=offOrig;
                            projParamsArrayHost[6*j+5]=source;
                        }   // END for (preparing params for kernel call)
                        
                        // Copy the prepared parameter arrays to constant memory to make it available for the kernel
                        hipMemcpyToSymbolAsync(HIP_SYMBOL(projSinCosArrayDev), projSinCosArrayHost, sizeof(float)*5*PROJ_PER_KERNEL,0,hipMemcpyHostToDevice,stream[dev*nStreamDevice]);
                        hipMemcpyToSymbolAsync(HIP_SYMBOL(projParamsArrayDev), projParamsArrayHost, sizeof(Point3D)*6*PROJ_PER_KERNEL,0,hipMemcpyHostToDevice,stream[dev*nStreamDevice]);
                        hipStreamSynchronize(stream[dev*nStreamDevice]);
                        
                        kernelPixelBackprojectionFDK<<<grid,block,0,stream[dev*nStreamDevice]>>>(geoArray[img_slice*deviceCount+dev],dimage[dev],i,proj_split_size[proj_block_split],texProj[(proj_block_split%2)*deviceCount+dev]);
                    }  // END for
                    //////////////////////////////////////////////////////////////////////////////////////
                    // END RB code, Main reconstruction loop: go through projections (rotation angles) and backproject
                    //////////////////////////////////////////////////////////////////////////////////////
                }// END for deviceCount
            } // END sub-split of current projection chunk
            
            for (dev = 0; dev < deviceCount; dev++){
                hipSetDevice(dev);
                hipDeviceSynchronize();
            }
            
        } // END projection splits
        
       
        // Now we need to take the image out of the GPU
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            // We do not need to sycnronize because the array dealocators already do.
            num_bytes_img_curr=(size_t)geoArray[img_slice*deviceCount+dev].nVoxelX*(size_t)geoArray[img_slice*deviceCount+dev].nVoxelY*(size_t)geoArray[img_slice*deviceCount+dev].nVoxelZ*sizeof(float);
            img_linear_idx_start=(size_t)geo.nVoxelX*(size_t)geo.nVoxelY*(size_t)geoArray[0].nVoxelZ*(size_t)(img_slice*deviceCount+dev);
            hipMemcpyAsync(&result[img_linear_idx_start], dimage[dev], num_bytes_img_curr, hipMemcpyDeviceToHost,stream[dev*nStreamDevice+1]);
        }
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipDeviceSynchronize();
            cudaCheckErrors("Main loop fail");
        }
        
    } // end image splits

    ///////// Cleaning:
    
    
    bool two_buffers_used=((((nalpha+split_projections-1)/split_projections)+PROJ_PER_KERNEL-1)/PROJ_PER_KERNEL)>1;
    for(unsigned int i=0; i<2;i++){ // 2 buffers (if needed, maybe only 1)
        if (!two_buffers_used && i==1)
            break;
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipDestroyTextureObject(texProj[i*deviceCount+dev]);
            hipFreeArray(d_cuArrTex[i*deviceCount+dev]);
        }
    }
    cudaCheckErrors("cudadestroy textures result fail");
    
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipFree(dimage[dev]);
    }
    hipHostFree(projSinCosArrayHost);
    hipHostFree(projParamsArrayHost);
    free(partial_projection);
    free(proj_split_size);
    
    freeGeoArray(split_image*deviceCount,geoArray);
    
    if (isHostRegisterSupported & (split_image>1 |deviceCount>1)){
        hipHostUnregister(result);
    }
//     if (isHostRegisterSupported){
//         hipHostUnregister(projections);
//     }
//     
    
    for (int i = 0; i < nStreams; ++i)
        hipStreamDestroy(stream[i]);
    
    cudaCheckErrors("hipFree fail");
    
    hipDeviceReset(); // For the Nvidia Visual Profiler
    return 0;
    
}  // END voxel_backprojection
//
void checkDevices(void){
    // CODE assumes
    // 1.-All available devices are usable by this code
    // 2.-All available devices are equal, they are the same machine (warning thrown)
    int dev;
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    const int devicenamelength = 256;  // The length 256 is fixed by spec of hipDeviceProp_t::name
    char devicename[devicenamelength];
    hipDeviceProp_t deviceProp;
    for (dev = 0; dev < deviceCount; dev++) {
        hipSetDevice(dev);
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev>0){
            if (strcmp(devicename,deviceProp.name)!=0){
                mexWarnMsgIdAndTxt("Atb:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed. \n Siddon_projection.cu line 275.");
                break;
            }
        }
        memset(devicename, 0, devicenamelength);
        strcpy(devicename, deviceProp.name);
    }
}
void splitCTbackprojection(int deviceCount,Geometry geo,int nalpha, unsigned int* split_image, unsigned int * split_projections){
    
    
    // We don't know if the devices are being used. lets check that. and only use the amount of memory we need.
    
    size_t mem_GPU_global;
    checkFreeMemory(deviceCount,&mem_GPU_global);
    
    
    // Compute how much memory each of the relevant memory pieces need
    size_t mem_image=       (unsigned long long)geo.nVoxelX*(unsigned long long)geo.nVoxelY*(unsigned long long)geo.nVoxelZ*sizeof(float);
    size_t mem_proj=        (unsigned long long)geo.nDetecU*(unsigned long long)geo.nDetecV*sizeof(float);
    
    
    
    
    // Does everything fit in the GPU?
    
    if(mem_image/deviceCount+mem_proj*PROJ_PER_KERNEL*2<mem_GPU_global){
        // We only need to split if we have extra GPUs
        *split_image=1;
        *split_projections=1;
    }
    // We know we need to split, but:
    // Does all the image fit in the GPU, with some slack for a stack of projections??
    else
    {
        // As we can overlap memcpys from H2D of the projections, we should then minimize the amount of image splits.
        // Lets assume to start with that we only need 1 stack of PROJ_PER_KERNEL projections. The rest is for the image.
        size_t mem_free=mem_GPU_global-2*mem_proj*PROJ_PER_KERNEL;
        
        *split_image=(mem_image/deviceCount+mem_free-1)/mem_free;
        // Now knowing how many splits we have for images, we can recompute how many slices of projections actually
        // fit on the GPU. Must be more than 0 obviously.
        
        mem_free=mem_GPU_global-(mem_image/deviceCount)/(*split_image); // NOTE: There is some rounding error, but its in the order of bytes, and we have 5% of GPU free jsut in case. We are safe
        
        
        *split_projections=(mem_proj*PROJ_PER_KERNEL*2+mem_free-1)/mem_free;
        
    }
}


void CreateTexture(int num_devices, float* projectiondata,Geometry geo,hipArray** d_cuArrTex,unsigned int nangles, hipTextureObject_t *texImage,hipStream_t* stream,int nStreamDevice,bool allocate){
    //size_t size_image=geo.nVoxelX*geo.nVoxelY*geo.nVoxelZ;
    const hipExtent extent =make_hipExtent(geo.nDetecV, geo.nDetecU, nangles);
    if (allocate){
        for (unsigned int dev = 0; dev < num_devices; dev++){
            hipSetDevice(dev);
            
            //hipArray Descriptor
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
            //cuda Array
            hipMalloc3DArray(&d_cuArrTex[dev], &channelDesc, extent);
            
        }
    }
    for (unsigned int dev = 0; dev < num_devices; dev++){
        hipSetDevice(dev);
        hipMemcpy3DParms copyParams = {0};
        //Array creation
        copyParams.srcPtr   = make_hipPitchedPtr((void *)projectiondata, extent.width*sizeof(float), extent.width, extent.height);
        copyParams.dstArray = d_cuArrTex[dev];
        copyParams.extent   = extent;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3DAsync(&copyParams,stream[dev*nStreamDevice+1]);
    }

    //Array creation End
    for (unsigned int dev = 0; dev < num_devices; dev++){
        hipSetDevice(dev);
        hipResourceDesc    texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array  = d_cuArrTex[dev];
        hipTextureDesc     texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        texDescr.filterMode = hipFilterModeLinear;
        texDescr.addressMode[0] = hipAddressModeBorder;
        texDescr.addressMode[1] = hipAddressModeBorder;
        texDescr.addressMode[2] = hipAddressModeBorder;
        texDescr.readMode = hipReadModeElementType;
        hipCreateTextureObject(&texImage[dev], &texRes, &texDescr, NULL);
    }
}

//______________________________________________________________________________
//
//      Function:       createGeoArray
//
//      Description:    This code generates the geometries needed to split the image properly in
//                      cases where the entire image does not fit in the memory of the GPU
//______________________________________________________________________________

void createGeoArray(unsigned int image_splits, Geometry geo,Geometry* geoArray, unsigned int nangles){
    
    
    unsigned int  splitsize=(geo.nVoxelZ+image_splits-1)/image_splits;
    
    for(unsigned int sp=0;sp<image_splits;sp++){
        geoArray[sp]=geo;
        // All of them are splitsize, but the last one, possible
        geoArray[sp].nVoxelZ=((sp+1)*splitsize<geo.nVoxelZ)?  splitsize:  max(geo.nVoxelZ-splitsize*sp,0);
        geoArray[sp].sVoxelZ= geoArray[sp].nVoxelZ* geoArray[sp].dVoxelZ;
        
        // We need to redefine the offsets, as now each subimage is not aligned in the origin.
        geoArray[sp].offOrigZ=(float *)malloc(nangles*sizeof(float));
        for (unsigned int i=0;i<nangles;i++){
            geoArray[sp].offOrigZ[i]=geo.offOrigZ[i]-geo.sVoxelZ/2+sp*geoArray[0].sVoxelZ+geoArray[sp].sVoxelZ/2;
        }
    }
    
}
//______________________________________________________________________________
//
//      Function:       freeGeoArray
//
//      Description:    Frees the memory from the geometry array for multiGPU.
//______________________________________________________________________________
void freeGeoArray(unsigned int splits,Geometry* geoArray){
    for(unsigned int sp=0;sp<splits;sp++){
        free(geoArray[sp].offOrigZ);
    }
    free(geoArray);
}
//______________________________________________________________________________
//
//      Function:       computeDeltasCube
//
//      Description:    Computes relative increments for each projection (volume rotation).
//						Increments get passed to the backprojection kernel.
//______________________________________________________________________________

void computeDeltasCube(Geometry geo,int i, Point3D* xyzorigin, Point3D* deltaX, Point3D* deltaY, Point3D* deltaZ,Point3D* S)
{
    
    Point3D P, Px,Py,Pz;
    // Get coords of Img(0,0,0)
    P.x=-(geo.sVoxelX/2-geo.dVoxelX/2)+geo.offOrigX[i];
    P.y=-(geo.sVoxelY/2-geo.dVoxelY/2)+geo.offOrigY[i];
    P.z=-(geo.sVoxelZ/2-geo.dVoxelZ/2)+geo.offOrigZ[i];
    
    // Get coors from next voxel in each direction
    Px.x=P.x+geo.dVoxelX;      Py.x=P.x;                Pz.x=P.x;
    Px.y=P.y;                   Py.y=P.y+geo.dVoxelY;    Pz.y=P.y;
    Px.z=P.z;                   Py.z=P.z;                Pz.z=P.z+geo.dVoxelZ;
    
    
    
// Rotate image around X axis (this is equivalent of rotating the source and detector) RZ RY RZ
    
    eulerZYZT(geo,&P);
    eulerZYZT(geo,&Px);
    eulerZYZT(geo,&Py);
    eulerZYZT(geo,&Pz);
    
    
    
    //detector offset
    P.z =P.z-geo.offDetecV[i];            P.y =P.y-geo.offDetecU[i];
    Px.z =Px.z-geo.offDetecV[i];          Px.y =Px.y-geo.offDetecU[i];
    Py.z =Py.z-geo.offDetecV[i];          Py.y =Py.y-geo.offDetecU[i];
    Pz.z =Pz.z-geo.offDetecV[i];          Pz.y =Pz.y-geo.offDetecU[i];
    
    //Detector Roll pitch Yaw
    //
    //
    // first, we need to offset everything so (0,0,0) is the center of the detector
    // Only X is required for that
    P.x=P.x+(geo.DSD[i]-geo.DSO[i]);
    Px.x=Px.x+(geo.DSD[i]-geo.DSO[i]);
    Py.x=Py.x+(geo.DSD[i]-geo.DSO[i]);
    Pz.x=Pz.x+(geo.DSD[i]-geo.DSO[i]);
    rollPitchYawT(geo,i,&P);
    rollPitchYawT(geo,i,&Px);
    rollPitchYawT(geo,i,&Py);
    rollPitchYawT(geo,i,&Pz);
    
    P.x=P.x-(geo.DSD[i]-geo.DSO[i]);
    Px.x=Px.x-(geo.DSD[i]-geo.DSO[i]);
    Py.x=Py.x-(geo.DSD[i]-geo.DSO[i]);
    Pz.x=Pz.x-(geo.DSD[i]-geo.DSO[i]);
    //Done for P, now source
    Point3D source;
    source.x=geo.DSD[i]; //allready offseted for rotation
    source.y=-geo.offDetecU[i];
    source.z=-geo.offDetecV[i];
    rollPitchYawT(geo,i,&source);
    
    
    source.x=source.x-(geo.DSD[i]-geo.DSO[i]);//   source.y=source.y-auxOff.y;    source.z=source.z-auxOff.z;
    
//       mexPrintf("%f,%f,%f\n",source.x,source.y,source.z);
    // Scale coords so detector pixels are 1x1
    
    P.z =P.z /geo.dDetecV;                          P.y =P.y/geo.dDetecU;
    Px.z=Px.z/geo.dDetecV;                          Px.y=Px.y/geo.dDetecU;
    Py.z=Py.z/geo.dDetecV;                          Py.y=Py.y/geo.dDetecU;
    Pz.z=Pz.z/geo.dDetecV;                          Pz.y=Pz.y/geo.dDetecU;
    
    source.z=source.z/geo.dDetecV;                  source.y=source.y/geo.dDetecU;
    
    // get deltas of the changes in voxels
    deltaX->x=Px.x-P.x;   deltaX->y=Px.y-P.y;    deltaX->z=Px.z-P.z;
    deltaY->x=Py.x-P.x;   deltaY->y=Py.y-P.y;    deltaY->z=Py.z-P.z;
    deltaZ->x=Pz.x-P.x;   deltaZ->y=Pz.y-P.y;    deltaZ->z=Pz.z-P.z;
    
    
    *xyzorigin=P;
    *S=source;
}

void eulerZYZT(Geometry geo, Point3D* point){
    
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x = auxPoint.x*(cos(geo.psi)*cos(geo.theta)*cos(geo.alpha)-sin(geo.psi)*sin(geo.alpha))
    +auxPoint.y*(-cos(geo.psi)*cos(geo.theta)*sin(geo.alpha)-sin(geo.psi)*cos(geo.alpha))
    +auxPoint.z*cos(geo.psi)*sin(geo.theta);
    point->y = auxPoint.x*(sin(geo.psi)*cos(geo.theta)*cos(geo.alpha)+cos(geo.psi)*sin(geo.alpha))
    +auxPoint.y*(-sin(geo.psi)*cos(geo.theta)*sin(geo.alpha)+cos(geo.psi)*cos(geo.alpha))
    +auxPoint.z*sin(geo.psi)*sin(geo.theta);
    point->z =-auxPoint.x*sin(geo.theta)*cos(geo.alpha)
    +auxPoint.y*sin(geo.theta)*sin(geo.alpha)
    +auxPoint.z*cos(geo.theta);
}
void rollPitchYawT(Geometry geo,int i, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=cos(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +sin(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.y
            -sin(geo.dPitch[i])*auxPoint.z;
    
    
    point->y=(cos(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) - sin(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.x
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) + cos(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +cos(geo.dPitch[i])*sin(geo.dYaw[i])*auxPoint.z;
    
    
    point->z=(cos(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) + sin(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.x
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) - cos(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.y
            +cos(geo.dPitch[i])*cos(geo.dYaw[i])*auxPoint.z;
    
}
void checkFreeMemory(int deviceCount,size_t *mem_GPU_global){
    size_t memfree;
    size_t memtotal;
    
    for (int dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipMemGetInfo(&memfree,&memtotal);
        if(dev==0) *mem_GPU_global=memfree;
        if(memfree<memtotal/2){
            mexErrMsgIdAndTxt("tvDenoise:tvdenoising:GPU","One (or more) of your GPUs is being heavily used by another program (possibly graphics-based).\n Free the GPU to run TIGRE\n");
        }
        cudaCheckErrors("Check mem error");
        
        *mem_GPU_global=(memfree<*mem_GPU_global)?memfree:*mem_GPU_global;
    }
    *mem_GPU_global=(size_t)((double)*mem_GPU_global*0.95);
    
    //*mem_GPU_global= insert your known number here, in bytes.
}

