#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for convolution
 *
 * Applies the convolution filter in the Fourier space.
 * The filter should be given in the Fourier transformed form.
 *
 * CODE by       Tomoyuki SADAKANE
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */

#include "TIGRE_common.hpp"
#include "FbpFiltration.hpp"
#include <string>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("apply_filtration",hipGetErrorString(__err));\
        } \
} while (0)

void cudafftCheckError(hipfftResult_t fftResult, const std::string& rstrMsg) {
    std::string strError = "Unknown error";
    if (fftResult == HIPFFT_SUCCESS){ return; }
    else if (fftResult == HIPFFT_INVALID_PLAN  ) { strError = "The plan parameter is not a valid handle. Handle is not valid when the plan is locked."; }
    else if (fftResult == HIPFFT_ALLOC_FAILED  ) { strError = "The allocation of GPU resources for the plan failed."; }
    else if (fftResult == HIPFFT_INVALID_VALUE ) { strError = "One or more invalid parameters were passed to the API."; }
    else if (fftResult == HIPFFT_INTERNAL_ERROR) { strError = "An internal driver error was detected."; }
    else if (fftResult == HIPFFT_SETUP_FAILED  ) { strError = "The cuFFT library failed to initialize."; }
    else if (fftResult == HIPFFT_INVALID_SIZE  ) { strError = "The nx or batch parameter is not a supported size."; }
    mexPrintf("%s \n", rstrMsg.c_str());
    mexErrMsgIdAndTxt("ApplyFiltration", strError.c_str());
}

__global__ void ApplyFilter(hipfftComplex* pcfInOut, size_t uiULen, size_t uiVLen, float* pfFilter, float fULInv) {

    size_t uiU = threadIdx.x + blockIdx.x * blockDim.x;
    size_t uiV = threadIdx.y + blockIdx.y * blockDim.y;
    if (uiV >= uiVLen || uiU >= uiULen) {
        return;
    }
    pcfInOut[uiU+uiULen*uiV].x *= pfFilter[uiU]*fULInv;
    pcfInOut[uiU+uiULen*uiV].y *= pfFilter[uiU]*fULInv;
}

//! Apply filter in the Fourier space
void apply_filtration(const float* pfIn, size_t uiULen, size_t uiVLen, const float* pfFilter, float* pfOut, const GpuIds& gpuids){
    // Prepare for MultiGPU
    int deviceCount = gpuids.GetLength();
    cudaCheckErrors("Device query fail");
    if (deviceCount == 0) {
        mexErrMsgIdAndTxt("apply_filtration","There are no available device(s) that support CUDA\n");
    }
    //
    // CODE assumes
    // 1.-All available devices are usable by this code
    // 2.-All available devices are equal, they are the same machine (warning thrown)
    // Check the available devices, and if they are the same
    if (!gpuids.AreEqualDevices()) {
        mexWarnMsgIdAndTxt("apply_filtration","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed.");
    }
    // USE THE FIRST GPU ONLY!!!!!!!!!!!!!!!!! 
    hipSetDevice(gpuids[0]);

    const size_t uiLen = uiULen * uiVLen;
    hipfftComplex* h_pcfInOut = (hipfftComplex*)malloc(uiLen*sizeof(hipfftComplex));

    if (!h_pcfInOut) {
        mexErrMsgIdAndTxt("ApplyFiltration", "apply_filtration fail hipHostMalloc 1");
    }
    for (int iV = 0; iV < uiVLen; ++iV) {
        for (int iU = 0; iU < uiULen; ++iU) {
            h_pcfInOut[iU+uiULen*iV] = hipfftComplex{pfIn[iU+uiULen*iV], 0};
        }
    }

    const float fULInv = 1./uiULen;

	hipfftHandle cudafftPlan;
    const int iBatch = uiVLen;
	hipfftResult_t fftresult;
    fftresult = hipfftPlan1d(&cudafftPlan, uiULen, HIPFFT_C2C, iBatch);
    cudafftCheckError(fftresult, "apply_filtration fail hipfftPlan1d 1");
    fftresult = hipfftPlan1d(&cudafftPlan, uiULen, HIPFFT_C2C, iBatch);
    cudafftCheckError(fftresult, "apply_filtration fail hipfftPlan1d 2");

    float*        d_pfFilter = nullptr;
    hipfftComplex* d_pcfInOut = nullptr;
    hipMalloc((void **)&d_pcfInOut, uiLen * sizeof(hipfftComplex));
    cudaCheckErrors("apply_filtration fail hipMalloc 1");
    hipMalloc((void **)&d_pfFilter, uiULen * sizeof(float));
    cudaCheckErrors("apply_filtration fail hipMalloc 2");
    hipMemcpy(d_pcfInOut, h_pcfInOut, uiLen* sizeof(hipfftComplex), hipMemcpyHostToDevice);
    cudaCheckErrors("apply_filtration fail hipMemcpy 1");
    hipMemcpy(d_pfFilter, pfFilter, uiULen * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("apply_filtration fail hipMemcpy 2");

    {
        const int divU = 128;//PIXEL_SIZE_BLOCK;
        const int divV = 1;//PIXEL_SIZE_BLOCK;
        dim3 grid((uiULen+divU-1)/divU,(uiVLen+divV-1)/divV,1);
        dim3 block(divU,divV,1);
        hipfftSetStream(cudafftPlan, 0);
        fftresult = hipfftExecC2C (cudafftPlan, d_pcfInOut, d_pcfInOut, HIPFFT_FORWARD);
        cudafftCheckError(fftresult, "apply_filtration fail hipfftExecC2C HIPFFT_FORWARD");
        ApplyFilter<<<grid, block>>>(d_pcfInOut, uiULen, uiVLen, d_pfFilter, fULInv);// Kernel d_pcfInOut = d_pcfInOut * pfFilter / uiULen
        fftresult = hipfftExecC2C (cudafftPlan, d_pcfInOut, d_pcfInOut, HIPFFT_BACKWARD);
        cudafftCheckError(fftresult, "apply_filtration fail hipfftExecC2C HIPFFT_BACKWARD");
    }

    hipMemcpy(h_pcfInOut, d_pcfInOut, uiLen*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    cudaCheckErrors("apply_filtration fail hipMemcpy 3");

    hipFree(d_pcfInOut); d_pcfInOut = nullptr;
    hipFree(d_pfFilter); d_pfFilter = nullptr;

    hipfftSetStream(cudafftPlan, 0);
    hipfftDestroy(cudafftPlan);

    for (int iV = 0; iV < uiVLen; ++iV) {
        for (int iU = 0; iU < uiULen; ++iU) {
            pfOut[iU+uiULen*iV] = h_pcfInOut[iU+uiULen*iV].x;
        }
    }
    free(h_pcfInOut); h_pcfInOut = nullptr;
}
