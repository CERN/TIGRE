#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for convolution
 *
 * Applies the convolution filter in the Fourier space.
 * The filter should be given in the Fourier transformed form.
 *
 * CODE by       Tomoyuki SADAKANE
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */

#include "TIGRE_common.hpp"
#include "FbpFiltration.hpp"
#include <string>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                mexErrMsgIdAndTxt("apply_filtration",hipGetErrorString(__err));\
        } \
} while (0)

void cudafftCheckError(hipfftResult_t fftResult, const std::string& rstrMsg) {
    std::string strError = "Unknown error";
    if (fftResult == HIPFFT_SUCCESS){ return; }
    else if (fftResult == HIPFFT_INVALID_PLAN  ) { strError = "The plan parameter is not a valid handle. Handle is not valid when the plan is locked."; }
    else if (fftResult == HIPFFT_ALLOC_FAILED  ) { strError = "The allocation of GPU resources for the plan failed."; }
    else if (fftResult == HIPFFT_INVALID_VALUE ) { strError = "One or more invalid parameters were passed to the API."; }
    else if (fftResult == HIPFFT_INTERNAL_ERROR) { strError = "An internal driver error was detected."; }
    else if (fftResult == HIPFFT_SETUP_FAILED  ) { strError = "The cuFFT library failed to initialize."; }
    else if (fftResult == HIPFFT_INVALID_SIZE  ) { strError = "The nx or batch parameter is not a supported size."; }
    mexPrintf("%s \n", rstrMsg.c_str());
    mexErrMsgIdAndTxt("ApplyFiltration", strError.c_str());
}

__global__ void ApplyFilter(hipfftComplex* pcfInOut, size_t uiULen, size_t uiVLen, float* pfFilter, float fULInv) {

    size_t uiU = threadIdx.x + blockIdx.x * blockDim.x;
    size_t uiV = threadIdx.y + blockIdx.y * blockDim.y;
    if (uiV >= uiVLen || uiU >= uiULen) {
        return;
    }
    pcfInOut[uiU+uiULen*uiV].x *= pfFilter[uiU]*fULInv;
    pcfInOut[uiU+uiULen*uiV].y *= pfFilter[uiU]*fULInv;
}

//! Apply filter in the Fourier space
void apply_filtration (const float* pfIn, size_t uiULen, size_t uiVLen, const float* pfFilter, float* pfOut, const GpuIds& gpuids) {
    // Prepare for MultiGPU
    int deviceCount = gpuids.GetLength();
    cudaCheckErrors("Device query fail");
    if (deviceCount == 0) {
        mexErrMsgIdAndTxt("apply_filtration","There are no available device(s) that support CUDA\n");
    }
    //
    // CODE assumes
    // 1.-All available devices are usable by this code
    // 2.-All available devices are equal, they are the same machine (warning thrown)
    // Check the available devices, and if they are the same
    if (!gpuids.AreEqualDevices()) {
        mexWarnMsgIdAndTxt("apply_filtration","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed.");
    }
    // USE THE FIRST GPU ONLY!!!!!!!!!!!!!!!!! 
    hipSetDevice(gpuids[0]);

    const size_t uiLen = uiULen * uiVLen;
    const float fULInv = 1./uiULen;

    float* d_pfInOut = nullptr;
    hipMalloc((void **)&d_pfInOut, uiLen * sizeof(float));
    cudaCheckErrors("apply_filtration fail hipMalloc 1");
    hipMemcpy(d_pfInOut, pfIn, uiLen* sizeof(float), hipMemcpyHostToDevice); // Sync only. pfIn is not pinned.
    cudaCheckErrors("apply_filtration fail hipMemcpy 1");

    size_t uiBufferSize = (uiULen+1)/2+1;    // Buffer size for R2C. See https://docs.nvidia.com/cuda/cufft/

	hipfftHandle cudafftPlanFwd;
	hipfftHandle cudafftPlanInv;
    const int iBatch = uiVLen;
	hipfftResult_t fftresult;
    fftresult = hipfftPlan1d(&cudafftPlanFwd, uiULen, HIPFFT_R2C, iBatch);
    cudafftCheckError(fftresult, "apply_filtration fail hipfftPlan1d 1");
    fftresult = hipfftPlan1d(&cudafftPlanInv, uiULen, HIPFFT_C2R, iBatch);
    cudafftCheckError(fftresult, "apply_filtration fail hipfftPlan1d 2");

    float* d_pfFilter = nullptr;
    hipMalloc((void **)&d_pfFilter, uiULen * sizeof(float));
    cudaCheckErrors("apply_filtration fail hipMalloc 2");
    hipMemcpy(d_pfFilter, pfFilter, uiULen * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("apply_filtration fail hipMemcpy 2");

    hipfftComplex* d_pcfWork = nullptr;
    hipMalloc((void **)&d_pcfWork, uiBufferSize * uiVLen*sizeof(hipfftComplex));
    cudaCheckErrors("apply_filtration fail hipMalloc 3");

    {
        const int divU = 128;//PIXEL_SIZE_BLOCK;
        const int divV = 1;//PIXEL_SIZE_BLOCK;
        dim3 grid((uiULen+divU-1)/divU,(uiVLen+divV-1)/divV,1);
        dim3 block(divU,divV,1);
        hipfftSetStream(cudafftPlanFwd, 0);
        hipfftSetStream(cudafftPlanInv, 0);
        fftresult = hipfftExecR2C (cudafftPlanFwd, d_pfInOut, d_pcfWork);
        cudafftCheckError(fftresult, "apply_filtration fail hipfftExecR2C");
        ApplyFilter<<<grid, block>>>(d_pcfWork, uiBufferSize, uiVLen, d_pfFilter, fULInv);// Kernel d_pcfInOut = d_pcfInOut * pfFilter / uiULen
        fftresult = hipfftExecC2R (cudafftPlanInv, d_pcfWork, d_pfInOut);
        cudafftCheckError(fftresult, "apply_filtration fail hipfftExecC2R");
    }
    hipMemcpy(pfOut, d_pfInOut, uiLen*sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("apply_filtration fail hipMemcpy 3");

    hipFree(d_pcfWork); d_pcfWork = nullptr;
    hipFree(d_pfInOut); d_pfInOut = nullptr;
    hipFree(d_pfFilter); d_pfFilter = nullptr;
    hipfftDestroy(cudafftPlanFwd);
    hipfftDestroy(cudafftPlanInv);
}


//! Apply filter in the Fourier space
void apply_filtration2 (const float* pfInAll, size_t uiOffset, size_t uiULen, size_t uiBatch, const float* pfFilter, size_t uiFLen, float fScale, float* pfOut, const GpuIds& gpuids) {
    // Prepare for MultiGPU
    int deviceCount = gpuids.GetLength();
    cudaCheckErrors("Device query fail");
    if (deviceCount == 0) {
        mexErrMsgIdAndTxt("apply_filtration","There are no available device(s) that support CUDA\n");
    }
    //
    // CODE assumes
    // 1.-All available devices are usable by this code
    // 2.-All available devices are equal, they are the same machine (warning thrown)
    // Check the available devices, and if they are the same
    if (!gpuids.AreEqualDevices()) {
        mexWarnMsgIdAndTxt("apply_filtration","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed.");
    }
    // USING THE FIRST GPU ONLY
    const float* pfIn = pfInAll+uiOffset;
    hipSetDevice(gpuids[0]);
    cudaCheckErrors("apply_filtration fail hipSetDevice");
    size_t uiPaddingLen = (uiFLen-uiULen) / 2;
    float* d_pfProjWide = nullptr;
    hipMalloc((void**)&d_pfProjWide, uiFLen*uiBatch*sizeof(float));
    cudaCheckErrors("apply_filtration fail hipMalloc wide");
    hipMemset(d_pfProjWide, 0, uiFLen*uiBatch*sizeof(float));
    cudaCheckErrors("apply_filtration fail hipMemset");
    hipMemcpy2D(&d_pfProjWide[uiPaddingLen], uiFLen*sizeof(float), pfIn, uiULen*sizeof(float), uiULen*sizeof(float), uiBatch, hipMemcpyHostToDevice);
    cudaCheckErrors("apply_filtration fail hipMemcpy2D");

    const float fFLInv = 1./uiFLen;

    size_t uiBufferSize = (uiFLen+1)/2+1;    // Buffer size for R2C. See https://docs.nvidia.com/cuda/cufft/

	hipfftHandle cudafftPlanFwd;
	hipfftHandle cudafftPlanInv;
	hipfftResult_t fftresult;
    fftresult = hipfftPlan1d(&cudafftPlanFwd, uiFLen, HIPFFT_R2C, uiBatch);
    cudafftCheckError(fftresult, "apply_filtration fail hipfftPlan1d 1");
    fftresult = hipfftPlan1d(&cudafftPlanInv, uiFLen, HIPFFT_C2R, uiBatch);
    cudafftCheckError(fftresult, "apply_filtration fail hipfftPlan1d 2");

    float* d_pfFilter = nullptr;
    hipMalloc((void **)&d_pfFilter, uiFLen * sizeof(float));
    cudaCheckErrors("apply_filtration fail hipMalloc 2");
    hipMemcpy(d_pfFilter, pfFilter, uiFLen * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("apply_filtration fail hipMemcpy 2");

    hipfftComplex* d_pcfWork = nullptr;
    hipMalloc((void **)&d_pcfWork, uiBufferSize * uiBatch*sizeof(hipfftComplex));
    cudaCheckErrors("apply_filtration fail hipMalloc 3");

    {
        const int divU = 128;//PIXEL_SIZE_BLOCK;
        const int divV = 1;//PIXEL_SIZE_BLOCK;
        dim3 grid((uiFLen+divU-1)/divU,(uiBatch+divV-1)/divV,1);
        dim3 block(divU,divV,1);
        hipfftSetStream(cudafftPlanFwd, 0);
        hipfftSetStream(cudafftPlanInv, 0);
        fftresult = hipfftExecR2C (cudafftPlanFwd, d_pfProjWide, d_pcfWork);
        cudafftCheckError(fftresult, "apply_filtration fail hipfftExecR2C");
        ApplyFilter<<<grid, block>>>(d_pcfWork, uiBufferSize, uiBatch, d_pfFilter, fFLInv*fScale);// Kernel d_pcfInOut = d_pcfInOut * pfFilter / uiFLen * 
        fftresult = hipfftExecC2R (cudafftPlanInv, d_pcfWork, d_pfProjWide);
        cudafftCheckError(fftresult, "apply_filtration fail hipfftExecC2R");
    }
    hipMemcpy2D(pfOut, uiULen*sizeof(float), &d_pfProjWide[uiPaddingLen], uiFLen*sizeof(float), uiULen*sizeof(float), uiBatch, hipMemcpyDeviceToHost);
    cudaCheckErrors("apply_filtration fail hipMemcpy 3");

    hipFree(d_pcfWork); d_pcfWork = nullptr;
    hipFree(d_pfProjWide); d_pfProjWide = nullptr;
    hipFree(d_pfFilter); d_pfFilter = nullptr;
    hipfftDestroy(cudafftPlanFwd);
    hipfftDestroy(cudafftPlanInv);
}